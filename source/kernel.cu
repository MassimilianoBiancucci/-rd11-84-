#include "hip/hip_runtime.h"
//////////////CUDA INCLUDES///////////////
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""

//////////////////////////////////////////

#include "stdafx.h"
#include <iostream>
#include <stdio.h>
#include <fstream>
#include <sstream>
#include <string>
#include <windows.h>
#include <limits>
#include <math.h>
#include <vector>
#include <list>
#include <time.h>
#include <algorithm>
#include <assert.h>

#undef max
#undef min

#define M_PI            3.14159265358979323846  /* pi */
#define maxOf(a, b) (((a) > (b)) ? a : b)

using namespace std;

struct Neuron;
typedef Neuron *ptNeuron;

struct arc {
	ptNeuron target = nullptr;
	float weight = 0;
	float oldDelta = 0;
	//bool enabled = true;
};

struct interArc {
	ptNeuron target = nullptr;
	ptNeuron base = nullptr;
};

struct Neuron {
	vector<arc> OutArcs; // to inizialize: = new vector<arc>(10);
	u_int numOutArcs = 0; // numero di archi in uscita
	u_int numInArcs = 0; //numero di archi in ingresso
	u_int layer = 0; //indice riga del neurone
	u_int column = 0; //indice della colonna del neurone
	float bayes = 0.01f; //peso del bayes
	float oldBayesDelta = 0; //ultima variazione del peso
	//vector<float> timeBayes; // vettore delle interconnessioni temporali
	vector<float> influenceInput; // vettore contenente la percentuale di influenza relativa ad ogni input
	vector<float> influenceOutput; // vettore contenente la percentuale di influenza relativa all'errore retropropagato da ogni output
	float output = 0; //potenziale attuale del neurone
	float absOutSum = 0; //somma in valore assoluto degli input del neurone
	float absDeltaSum = 0; //somma in valore assoluto delle variazioni dei pesi
	float BPerr = 0; //errore di retropropagazione
	int neurIdx = 0; //ogni neurone � contraddistinto da un indice unico che si riferisce alla sua posizione
};

struct Layer {
	vector<Neuron> Neurons;
	u_int numNeurons = 0;
};

struct Omap {
	float maxValue = 1;
	float minValue = 0;
};

struct conMap { // struttura necessaria per l'inserimento di un nuovo neurone
	u_int startLyr;
	u_int startCol;
	u_int arcRef;
	u_int targetLyr;
	u_int targetCol;
};

struct timeSeries {
	list<float> evento;
};

struct example {
	vector<float> input;
	vector<float> Doutput;
};

struct Dataset {
	vector<example> trainingSet;
	vector<example> validationSet;
	float triningErr = 0;
	float validationErr = 0;
};

class DatasetCore {
public:
	list<Dataset> Datasets;

	DatasetCore() {

	}
	////////////////////////////////////////////////MANIPOLAZIONE DATASET//////////////////////////////////////
	void readTimeSeriesCsv(string filename, int outStep, int inEx, float trainingPerc) {
		//outStep - rappresenta il numero di valori per esempio output
		//inEx - � il numero di esempi output precedenti che vengono passati in ogni esempio input
		//trainingPerc - � la percentuale di dataset da usare come trainingset
		cout << "caricamento del file " << filename << endl;
		ifstream file(filename + ".csv");
		stringstream stream;
		stream << file.rdbuf();
		string line, cell;
		int col = 0, row = -1;
		int fileRow = coutnRow(filename);
		list<timeSeries> esempi(fileRow);
		list<timeSeries>::iterator it = esempi.begin();

		while (getline(stream, line, '\n')) { //smista righe
			stringstream streamline(line);
			col = 0; row++;
			it->evento.resize(0);
			while (getline(streamline, cell, ';')) {
				//cout << stof(cell) << endl;
				it->evento.push_back(stof(cell));
			}
			it++;
		}

		Dataset newSet;
		example newEx;
		newEx.Doutput.resize(outStep);
		newEx.input.resize(inEx*outStep);
		int nEx = esempi.size() - inEx;
		int ntrainEx = (int)((trainingPerc / 100) * nEx);
		int nvalidateEx = nEx - ntrainEx;
		int sPos = 0, ePos = inEx, pos = 0; // il passo � outStep
		int inId = 0, outId = 0;
		for (int i = 0; i < nEx; i++) {
			pos = 0, inId = 0, outId = 0;

			for (list<timeSeries>::iterator p = esempi.begin(); p != esempi.end(); p++) {
				for (list<float>::iterator q = p->evento.begin(); q != p->evento.end(); q++) {
					if (pos >= sPos && pos <= ePos) {
						if (pos >= inEx + sPos) { // carico il vettore output
							newEx.Doutput[outId] = *q;
							outId++;
						}
						else { //carico il vettore di input
							newEx.input[inId] = *q;
							inId++;
						}
					}
				}
				pos++;
			}

			if (sPos < ntrainEx) { // carico il trining set
				newSet.trainingSet.push_back(newEx);
			}
			else { //carico il validationset
				newSet.validationSet.push_back(newEx);
			}
			//cout << sPos << endl;
			sPos++;
			ePos++;
		}
		Datasets.push_back(newSet);
	}
	vector<example> getDataset(int n, bool training = true) {
		list<Dataset>::iterator p = Datasets.begin();
		for (int k = 0; k < n; k++) p++;
		if (training == true) {
			return p->trainingSet;
		}
		else {
			return p->validationSet;
		}
	}
	////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////ALTRE FUNZIONI//////////////////////////////////////////////
	int coutnRow(string filename) {
		ifstream file(filename + ".csv");
		stringstream stream;
		stream << file.rdbuf();
		string line;
		int row = 0;
		while (getline(stream, line, '\n'))row++;
		return row;
	}
	////////////////////////////////////////////////////////////////////////////////////////////////////////////
};

class Network {

public:
	vector<Layer> Layers; // vettore di struct layer
	vector<example> examples; // vettore di esempi per l'apprendimento
	vector<Omap> map; // vettore contenente i valori di rimappatura del'output della rete

	string genoma = ""; // nome del file
	u_int nLayers = 0; // Layer nella rete compresi input output
	u_int nNeurons = 0; // numero totale neuroni nella rete
	u_int nArc = 0; //numero totale di tutti gli archi presenti nella rete

	Network(string filename) {
		genoma = filename;
	}

	////////////////////////////////////FUNZIONE COSTRUZIONE RETE DA FILE/////////////////////////////////////////////////////////
	void getNetParams() {
		ifstream file(genoma + ".txt");
		string line, segment, pice;
		int flag = 4; // 3- NL, 2-Ln, 1-NCON, 0-CON
		int pos;
		int lyr, neuronsInLyr, Nneuron, con; // numero layer, numero neuroni in dato layer, numero connessioni d'uscita in un dato neurone
		int Tlyr, Tneuron; // target layer, target neuron
		int conFlag; // 0 - inserisco la connessione, 1 - inserisco il peso
		float bayes;
		//file.open(filename + ".txt");
		stringstream stream;
		stream << file.rdbuf();

		if (file.is_open()) {
			//cout << "si e' apertpo" << endl;
			while (getline(stream, line, '\n')) { //smista righe
				stringstream streamline(line);
				pos = 0;
				while (getline(streamline, segment, ' ')) { // smista segmenti riga
															//cout << segment << endl;
					if (flag != 0 && pos == 0) {
						if (segment == "CON") {
							flag = 0;
						}
						else if (segment == "NCON") {
							flag = 1;
						}
						else if (segment == "L0") {
							flag = 2;
						}
						else if (segment == "NL") {
							flag = 3;
						}
					}

					if (flag == 0 && segment != "CON") { // dichiarazione delle connessioni
														 //cout << "flag0" << endl;
						if (pos == 0) {
							stringstream streamsegment(segment);
							getline(streamsegment, pice, '-'); // file syntax CON \n 0-0
							lyr = stoi(pice); // rimane fissato per tutta la riga
							getline(streamsegment, pice);
							Nneuron = stoi(pice);
							conFlag = 0;

						}
						else if (lyr != nLayers) {
							if (conFlag == 0) { //inizializzo la connessione

								stringstream streamsegment(segment);
								getline(streamsegment, pice, '-'); // file syntax CON \n 1-0
								Tlyr = stoi(pice);
								getline(streamsegment, pice, ' ');
								Tneuron = stoi(pice);
								Layers[lyr].Neurons[Nneuron].OutArcs[(pos - 1) / 2].target = &(Layers[Tlyr].Neurons[Tneuron]);
								Layers[Tlyr].Neurons[Tneuron].numInArcs++;
								conFlag = 1;

							}
							else { //inizzializzo il peso della connessione 

								Layers[lyr].Neurons[Nneuron].OutArcs[(pos - 2) / 2].weight = stof(segment);
								conFlag = 0;

							}
						}

					}
					else if (flag == 1 && segment != "NCON") { // dichiarazione numero connessioni
															   //cout << "flag1" << endl;
						if (pos == 0) {
							stringstream streamsegment(segment);
							getline(streamsegment, pice, '-'); // file syntax NCON \n 0-0_4
							lyr = stoi(pice);
							getline(streamsegment, pice, '_');
							Nneuron = stoi(pice);
							getline(streamsegment, pice, ':');
							con = stoi(pice);
							getline(streamsegment, pice, ' ');
							bayes = stof(pice);

							if (con >= 0) {
								Layers[lyr].Neurons[Nneuron].numOutArcs = con;
								nArc += con;
								Layers[lyr].Neurons[Nneuron].OutArcs.resize(con, arc());  //= new vector<arc>(con); // dichiarazione numero connessioni del neurone
								Layers[lyr].Neurons[Nneuron].layer = lyr;
								Layers[lyr].Neurons[Nneuron].column = Nneuron;
								Layers[lyr].Neurons[Nneuron].bayes = bayes;
							}
						}
					}
					else if (flag == 2) { // numero neuroni per layer
										  //cout << "flag2" << endl;
						if (pos == 0) {
							lyr = stoi(segment.erase(0, 1));
						}
						else {
							neuronsInLyr = stoi(segment);
							Layers[lyr].numNeurons = neuronsInLyr;
							nNeurons += neuronsInLyr;
							Layers[lyr].Neurons.resize(neuronsInLyr, Neuron());  // = new vector<Neuron>(neuronsInLyr); // dichiarazione numero di neuroni per layer
																				 //cout << segment << " " << lyr << endl;
						}

					}
					else if (flag == 3 && pos != 0) { // numero layer
													  //cout << "flag3" << endl;
						Layers.resize(stoi(segment), Layer()); //= new vector<Layer>(stoi(segment)); // dichiarazione numero di layer
						nLayers = stoi(segment);
						//cout << segment << endl;
					}
					pos++;
				}
			}

			for (int i = 0; i < Layers[nLayers - 1].numNeurons; i++) {
				Layers[nLayers - 1].Neurons[i].column = i;
				Layers[nLayers - 1].Neurons[i].layer = nLayers - 1;
			}

		}
		else {

			cout << genoma + ": errore nell'apertura o file non trovato" << endl;

		}
		file.close();
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////FUNZIONE COSTRUZIONE DATASET DA FILE//////////////////////////////////////////////////////
	void getDataset(string filename) {
		ifstream file(filename + ".txt");
		string line, segment, pice;
		int flag = 4; // 3- NE, 2-IN, 1-OUT, 0-ES
		int pos, esPos;
		int input, output, es; // numero layer, numero neuroni in dato layer, numero connessioni d'uscita in un dato neurone
		int IOFlag = 0; // 0 - input esempio, 1 - output esempio
						//file.open(filename + ".txt");
		stringstream stream;
		stream << file.rdbuf();

		if (file.is_open()) {
			//cout << "si e' apertpo" << endl;
			while (getline(stream, line, '\n')) { //smista righe
				stringstream streamline(line);
				pos = 0;
				while (getline(streamline, segment, ' ')) { // smista segmenti riga
															//cout << segment << endl;
					if (flag != 0 && pos == 0) {
						if (segment == "ES") {
							flag = 0;
						}
						else if (segment == "OUT") {
							flag = 1;
						}
						else if (segment == "IN") {
							flag = 2;
						}
						else if (segment == "NE") {
							flag = 3;
						}
					}
					if (flag == 0 && segment == "ES") { // caricamento degli esmpi
														//cout << "flag0" << endl;
						goto examplesLoading;
					}
					else if (flag == 1 && pos != 0) { // numero output
													  //cout << "flag1" << endl;
						output = stoi(segment);
						if (output != numNeurons(nLayers - 1)) { cout << "Il dataset deve avere lo stesso numero di output della rete!!" << endl; ClearDataset(); return; }
					}
					else if (flag == 2 && pos != 0) { // numero input
													  //cout << "flag2" << endl;
						input = stoi(segment);
						if (input != numNeurons(0)) { cout << "Il dataset deve avere lo stesso numero di input della rete!!" << endl; ClearDataset(); return; }
					}
					else if (flag == 3 && pos != 0) { // numero esempi
													  //cout << "flag3" << endl;
						examples.resize(stoi(segment), example()); //= new vector<Layer>(stoi(segment)); // dichiarazione numero di layer
																   //cout << segment << endl;
					}
					pos++;
				}
			}

		examplesLoading:
			esPos = 0;
			while (getline(stream, line, '\n')) { //smista righe
				stringstream streamline(line);
				pos = 0;
				if (!IOFlag) { // carico gli input
					examples[esPos].input.resize(input);
					while (getline(streamline, segment, ' ')) { // smista segmenti riga
						examples[esPos].input[pos] = stof(segment);
						pos++;
					}
					IOFlag = 1;
				}
				else { // carico gli output
					examples[esPos].Doutput.resize(output);
					while (getline(streamline, segment, ' ')) { // smista segmenti riga
						examples[esPos].Doutput[pos] = stof(segment);
						pos++;
					}
					IOFlag = 0;
					esPos++;
				}

			}

		}
		else {

			cout << genoma + ": errore nell'apertura o file non trovato" << endl;

		}
		file.close();

	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////SALVA RETE SU FILE///////////////////////////////////////////////////////////////
	void saveNet(string filename = "") {

		cout << "saving  net.. " << endl;
		ofstream file;

		if (filename != "") {
			file.open(filename + ".txt");
		}
		else {
			file.open(genoma + ".txt");
		}
		file << "NL " << nLayers << '\n';

		for (int i = 0; i < nLayers; i++) file << "L" << i << " " << Layers[i].numNeurons << '\n';

		file << "NCON" << '\n';
		for (int b = 0; b < nLayers; b++) {
			for (int d = 0; d < Layers[b].numNeurons; d++) {
				file << b << "-" << d << "_" << Layers[b].Neurons[d].numOutArcs << ":" << Layers[b].Neurons[d].bayes << '\n';
			}
		}

		file << "CON";
		for (int b = 0; b < nLayers; b++) {
			//cout << "salvataggio connessioni layer:" << b << endl;
			for (int d = 0; d < Layers[b].numNeurons; d++) {
				file << '\n';
				file << b << "-" << d;

				for (int c = 0; c < Layers[b].Neurons[d].numOutArcs; c++) {
					file << " ";
					file << Layers[b].Neurons[d].OutArcs[c].target->layer << "-" << Layers[b].Neurons[d].OutArcs[c].target->column << " " << Layers[b].Neurons[d].OutArcs[c].weight;
				}
			}
		}
		file.close();
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////SALVA DATASET SU FILE///////////////////////////////////////////////////////////////
	void saveDataset(string filename) {
		ofstream file(filename + ".txt");
		file << "NE " << examples.size() << '\n';
		file << "IN " << Layers[0].numNeurons << '\n';
		file << "OUT " << Layers[nLayers - 1].numNeurons << '\n';
		file << "ES" << '\n';
		for (int i = 0; i < examples.size(); i++) {
			for (int j = 0; j < examples[i].input.size(); j++) { // salvo gli input 
				file << examples[i].input[j];
				if (j == examples[i].input.size() - 1) { file << '\n'; }
				else { file << " "; }
			}
			for (int j = 0; j < examples[i].Doutput.size(); j++) { //salvo gli output
				file << examples[i].Doutput[j];
				if (j == examples[i].Doutput.size() - 1) { file << '\n'; }
				else { file << " "; }
			}
		}

	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////// FUNZIONI MODIFICA RETE//////////////////////////////////////////////////////////
	void deleteArc(int Nlayer, int Ncolumn, int targetLayer, int targetColumn) {
		for (int i = 0; i < Layers[Nlayer].Neurons[Ncolumn].numOutArcs; i++) {
			if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->layer == targetLayer && Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->column == targetColumn) {

				//modifico i parametri della rete relazionati a tale arco
				Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->numInArcs--;
				if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->influenceInput.size() > 0)Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->influenceInput.pop_back();
				if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->influenceOutput.size() > 0)Layers[Nlayer].Neurons[Ncolumn].influenceOutput.pop_back();

				if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->numInArcs == 0 && Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->layer != 0) {
					deleteNeuron(Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->layer, Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->column);
				}
				else {
					Layers[Nlayer].Neurons[Ncolumn].OutArcs.erase(Layers[Nlayer].Neurons[Ncolumn].OutArcs.begin() + i);
					Layers[Nlayer].Neurons[Ncolumn].numOutArcs--;
				}

				return;
			}
		}
		cout << "errore: arco (" << Nlayer << "-" << Ncolumn << ") -> (" << targetLayer << "-" << targetColumn << ") non trovato, eliinazione fallita" << endl;
	}
	void deleteArcByRef(int Nlayer, int Ncolumn, arc *targetArc) { //elimina un arco con gli indici della base e il puntatore alla connsessione

		//modifico i parametri della rete relazionati a tale arco
		targetArc->target->numInArcs--;
		if (targetArc->target->influenceInput.size() > 0)targetArc->target->influenceInput.pop_back();
		if (Layers[Nlayer].Neurons[Ncolumn].influenceOutput.size() > 0)Layers[Nlayer].Neurons[Ncolumn].influenceOutput.pop_back();

		if (targetArc->target->numInArcs == 0) {
			deleteNeuron(targetArc->target->layer, targetArc->target->column);
		}
		else {
			Layers[Nlayer].Neurons[Ncolumn].OutArcs.erase(Layers[Nlayer].Neurons[Ncolumn].OutArcs.begin() + getOutConTargetID(getTarget(Nlayer, Ncolumn), targetArc->target));
			Layers[Nlayer].Neurons[Ncolumn].numOutArcs--;
		}

	}
	void deleteNeuron(int Nlayer, int Ncolumn) {
		// elimino l'arco dai neuroni che puntano al nodo da eliminare e
		// decremento il numero di archi in output hai neuroni con come target il neurone da eliminare
		for (int i = 0; i < Nlayer; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				for (int k = 0; k < Layers[i].Neurons[j].numOutArcs; k++) {
					if (Layers[i].Neurons[j].OutArcs[k].target->layer == Nlayer && Layers[i].Neurons[j].OutArcs[k].target->column == Ncolumn) {
						deleteArcByRef(i, j, &Layers[i].Neurons[j].OutArcs[k]);
					}
				}
			}
		}
		// eliminio gli archi del neurone da eliminare
		for (int i = 0; i < Layers[Nlayer].Neurons[Ncolumn].numOutArcs; i++) {
			deleteArcByRef(Nlayer, Ncolumn, &Layers[Nlayer].Neurons[Ncolumn].OutArcs[i]);
		}

		// decremento l'indice colonna dei neuroni successivi nello stesso layer
		for (int i = Ncolumn + 1; i < Layers[Nlayer].numNeurons; i++) Layers[Nlayer].Neurons[i].column--;

		// elimino il neurone
		Layers[Nlayer].Neurons.erase(Layers[Nlayer].Neurons.begin() + Ncolumn);
		Layers[Nlayer].numNeurons--;
		nNeurons--;
	}
	void addArc(int Nlayer, int Ncolumn, int targetLayer, int targetColumn) {
		arc newArc; newArc.target = &(Layers[targetLayer].Neurons[targetColumn]); newArc.weight = 0.01f;
		Layers[targetLayer].Neurons[targetColumn].numInArcs++;
		Layers[Nlayer].Neurons[Ncolumn].OutArcs.push_back(newArc);
		Layers[Nlayer].Neurons[Ncolumn].numOutArcs++;
	}
	void addNeuron(int Nlayer, float inConFill, float outConFill) {
		//init new neuron
		Neuron newNeur;
		newNeur.layer = Nlayer;
		newNeur.column = Layers[Nlayer].numNeurons; // column start at 0 , numNeurons start at 1
		int nBackNeurons = 0; for (int i = 0; i < Nlayer; i++) nBackNeurons += Layers[i].numNeurons;
		int nFrontNeurons = 0; for (int i = Nlayer + 1; i < nLayers; i++) nFrontNeurons += Layers[i].numNeurons;
		newNeur.numOutArcs = (int)(outConFill*nFrontNeurons);
		newNeur.numInArcs = (int)(inConFill*nBackNeurons);
		newNeur.OutArcs.resize(newNeur.numOutArcs);

		//bind neuron with neuron of front layers (OUTPUT CONNECTIONS)
		int x, y = 0;
		vector<u_int> OutArcAcces = casualVector(newNeur.numOutArcs);
		vector<u_int> NetAcces = casualVector(nFrontNeurons);
		while (y < OutArcAcces.size()) {
			x = 0;
			for (int i = Nlayer + 1; i < nLayers; i++) {
				for (int j = 0; j < Layers[i].numNeurons; j++) {
					if (y > OutArcAcces.size() - 1) continue;
					if (NetAcces[y] == x) {
						newNeur.OutArcs[OutArcAcces[y]].target = &(Layers[i].Neurons[j]);
						newNeur.OutArcs[OutArcAcces[y]].weight = 0.01f;
						y++;
					}
					x++;
				}
			}
		}
		cout << "connessioni output create" << endl << endl;
		WeightsStamp("w");

		// inserisco il neurone nella rete
		//NOTA push_back() reinizializa il vettore modificandi i puntatori alle celle, tutte le connessioni al layer che viene aggiornato vengono perse
		vector<conMap> connections = saveConsTowardsLyr(Nlayer); // salvo le connessioni al layer
		Layers[Nlayer].Neurons.push_back(newNeur); // immetto il nuovo neurone (i riferimenti del vettore cambiano)
		Layers[Nlayer].numNeurons++;
		loadConsTowardsLyr(connections); // aggiorno i puntatori collegati ai neuroni di questo vettore


		WeightsStamp("w");
		cout << "neurone inserito" << endl << endl;

		// bind neuron with back layers (INPUT CONNECTIONS)	
		NetAcces = casualVector(nBackNeurons);
		arc newArc; newArc.weight = 0.01f;
		y = 0;
		while (y < newNeur.numInArcs) {
			x = 0;
			for (int i = 0; i < Nlayer; i++) {
				for (int j = 0; j < Layers[i].numNeurons; j++) {
					if (y > newNeur.numInArcs - 1) continue;
					if (NetAcces[y] == x) {
						newArc.target = &(Layers[Nlayer].Neurons[Layers[Nlayer].numNeurons - 1]);
						//Layers[Nlayer].Neurons[Layers[Nlayer].numNeurons - 1].numInArcs++;
						Layers[i].Neurons[j].OutArcs.push_back(newArc);
						Layers[i].Neurons[j].numOutArcs++;
						cout << "BIND:" << y << " (" << i << "-" << j << ")  ->  (" << Nlayer << "-" << Layers[Nlayer].numNeurons - 1 << ") " << endl;
						WeightsStamp("w");
						y++;
					}
					x++;
				}
			}
		}
		nNeurons++;
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////FUNZIONI VARIE///////////////////////////////////////////////////////////
	float DsigOut(int Layer, int Neuron) {
		//derivata della funzione sigmoide (Y*(1-Y)) 
		//RICORDA di aggiungere k se implementi la sensibilit� della sigmoide
		return Layers[Layer].Neurons[Neuron].output*(1 - Layers[Layer].Neurons[Neuron].output);
	}
	float sigmoid(float x) { return 1 / (1 + exp(-x)); } // Sigmoide
	float logit(float x) { return log(x / (1 - x)); } // funzione sigmoide inversa 
	float gaussian(float x, float mu, float var) { return (1 / (var*sqrt(2 * M_PI)))*exp(-((pow(x - mu, 2.0)) / (2 * var))); } // Gaussiana(float x, float mu, float var)
	void WeightsStamp(string mode = "a") {
		//m - stampa le medie dei pesi di ogni layer
		//a - stampa tutti i pesi della rete con alcuni parametri di apprendimento
		//w - stampa tutti i pesi con il riferimento riga colonna al target
		//fc - stampa le medie dei gruppi di pesi tra due layer 
		if (mode == "m") { // medie dei layer
			float mean;
			int x;
			for (int i = 0; i < numLayers() - 1; i++) {
				x = 0;
				mean = 0;
				for (int c = 0; c < numNeurons(i); c++) {
					for (int d = 0; d < numCon(i, c); d++) {
						mean += getWeight(i, c, d);
						x++;
					}
				}
				mean /= x;
				cout << "Layer " << i << " weights:" << mean << endl;
			}
		}
		else if (mode == "a") { // stampa tutti i pesi e i parametri dei neuroni
			for (int i = 0; i < numLayers() - 1; i++) {
				for (int c = 0; c < numNeurons(i); c++) {
					cout << "(" << i << "-" << c << ")  output: " << getOutput(i, c) << "  BPerr: " << getBPerr(i, c) << endl;
					for (int d = 0; d < numCon(i, c); d++) {
						cout << getWeight(i, c, d) << " (" << getDeltaWeight(i, c, d) << ")  ";
					}
					cout << endl;
				}
			}
		}
		else if (mode == "w") { // stampa tutti i pesi con il riferimento al target
			for (int i = 0; i < numLayers() - 1; i++) {
				for (int c = 0; c < numNeurons(i); c++) {
					cout << "(" << i << "-" << c << ")" << "Con: " << numCon(i, c) << "  InCon: " << numInCon(i, c) << endl;
					for (int d = 0; d < numCon(i, c); d++) {
						cout << getWeight(i, c, d) << " (" << getConTargetLyr(i, c, d) << "," << getConTargetCol(i, c, d) << ")  ";
					}
					cout << endl;
				}
			}
		}
		else if (mode == "fc") { // stampa le medie dei pesi tra layer e layer
			vector<float> wLyr(nLayers - 1);
			vector<int> nwLyr(nLayers - 1);
			float wl = 0;
			for (int i = 0; i < nLayers - 1; i++) {
				for (int j = 0; j < Layers[i].numNeurons; j++) {
					for (int k = 0; k < Layers[i].Neurons[j].numOutArcs; k++) {
						wLyr[Layers[i].Neurons[j].OutArcs[k].target->layer - 1] += Layers[i].Neurons[j].OutArcs[k].weight;
						nwLyr[Layers[i].Neurons[j].OutArcs[k].target->layer - 1]++;
					}
				}
				cout << "Layer " << i << " weights: ";
				for (int j = 0; j < wLyr.size(); j++) {
					wLyr[j] /= nwLyr[j];
					if (j + 1 > i)cout << "(" << i << ", " << j + 1 << ") [" << nwLyr[j] << "] " << wLyr[j] << "  ";
				}
				cout << endl;
				fill(wLyr.begin(), wLyr.end(), 0);
				fill(nwLyr.begin(), nwLyr.end(), 0);
			}
		}
		else {
			cout << "WeightsStamp() argument error!" << endl;
		}
		cout << endl;
	}
	void sigLayer(int lyr) {  // applica la sigmoide a tutti i campi output dei neuroni nel layer specificato
		for (int i = 0; i < Layers[lyr].numNeurons; i++) {
			Layers[lyr].Neurons[i].output = sigmoid(Layers[lyr].Neurons[i].output);
		}
	}
	void bayesLayer(int lyr, bool absSum = false) {//applica il bayes all'output di ogni neurone del dato layer
		for (int i = 0; i < Layers[lyr].numNeurons; i++) {
			Layers[lyr].Neurons[i].output += Layers[lyr].Neurons[i].bayes;
		}
		// se la variabile absSum � true sommo il bayes in valore assoluto alla variabile absOutSum
		if (absSum == true) { for (int i = 0; i < Layers[lyr].numNeurons; i++) { Layers[lyr].Neurons[i].absOutSum += abs(Layers[lyr].Neurons[i].bayes); } }
	}
	void resetPotential() {
		// esegue il reset del potenziale di tutti i neuroni della rete
		for (int i = 0; i < nLayers; i++) {
			for (int j = 0; j < numNeurons(i); j++) {
				Layers[i].Neurons[j].output = 0;
			}
		}
	}
	void resetAbsSumPotenzial() {
		// esegue il reset della sommatoria di ogni input in valore assoluto di ogni neurone
		for (int i = 0; i < nLayers; i++) {
			for (int j = 0; j < numNeurons(i); j++) {
				Layers[i].Neurons[j].absOutSum = 0;
			}
		}
	}
	void resetAbsSumDelta() {
		// esegue il reset della sommatoria di ogni input in valore assoluto di ogni neurone
		for (int i = 0; i < nLayers; i++) {
			for (int j = 0; j < numNeurons(i); j++) {
				Layers[i].Neurons[j].absDeltaSum = 0;
			}
		}
	}
	void resetBPerr() { // esegue il reset dell'errore retropropagato in ogni neurone
		for (int i = 0; i < nLayers; i++) {
			for (int j = 0; j < numNeurons(i); j++) {
				Layers[i].Neurons[j].BPerr = 0;
			}
		}
	}
	void resetNeuronsID() {
		for (int i = 0; i < nLayers; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				Layers[i].Neurons[j].layer = i;
				Layers[i].Neurons[j].column = j;
			}
		}
	}
	vector<conMap> saveConsTowardsLyr(int Layer) { // salva su vettore i riferimenti numerici delle connesioni verso il layer specificato
		vector<conMap> connections;
		conMap con;
		for (int i = 0; i < Layer; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				for (int k = 0; k < Layers[i].Neurons[j].numOutArcs; k++) {
					if (Layers[i].Neurons[j].OutArcs[k].target->layer == Layer) {
						con.startCol = j;
						con.startLyr = i;
						con.arcRef = k;
						con.targetCol = Layers[i].Neurons[j].OutArcs[k].target->column;
						con.targetLyr = Layers[i].Neurons[j].OutArcs[k].target->layer;
						connections.push_back(con);
					}
				}
			}
		}
		return connections;
	}
	void loadConsTowardsLyr(vector<conMap> con) { // ricarica i riferimenti numerici delle connesioni verso il layer specificato
		for (int i = 0; i < con.size(); i++) {
			Layers[con[i].startLyr].Neurons[con[i].startCol].OutArcs[con[i].arcRef].target = getTarget(con[i].targetLyr, con[i].targetCol);
		}
	}
	void ClearDataset() { examples.clear(); }
	void genTestDataset(int nExe, int nIn, int nOut, float step, int type, float offset) { //generazione di una serie storica del seno (DEBUG)
		cout << "creating examples.." << endl;
		examples.resize(nExe);
		float x = 0, x2 = 0;
		for (int i = 0; i < nExe; i++) {
			examples[i].input.resize(nIn);
			examples[i].Doutput.resize(nOut);
			switch (type) {
			case 0: // debug dataset
				for (int j = nIn - 1; j >= 0; j--) {
					examples[i].input[j] = 1 + offset;
				}
				for (int j = 0; j < nOut; j++) {
					examples[i].Doutput[j] = 1 + offset;
				}
				break;
			case 1: // funzione predizione
				for (int j = 0; j < nIn; j++) {
					examples[i].input[j] = sin(x2) + offset;
					x2 += step;
				}
				for (int j = 0; j < nOut; j++) {
					examples[i].Doutput[j] = sin(x2) + offset;
					x2 += step;
				}
				x += step;
				x2 = x;
				break;
			case 2: // funzione uguale
				for (int j = 0; j < nIn; j++) {
					examples[i].input[j] = sin(x2) + offset;
					x2 += step;
				}
				x2 = x;
				for (int j = 0; j < nOut; j++) {
					examples[i].Doutput[j] = sin(x2) + offset;
					x2 += step;
				}
				x += step;
				x2 = x;
				break;
			case 3: // funzione specchio
				for (int j = nIn - 1; j >= 0; j--) {
					examples[i].input[j] = sin(x2) + offset;
					x2 += step;
				}
				x2 = x;
				for (int j = 0; j < nOut; j++) {
					examples[i].Doutput[j] = sin(x2) + offset;
					x2 += step;
				}
				x += step;
				x2 = x;
				break;
			}
		}
	}
	void setNetMap(float max, float min) {
		map.resize(Layers[nLayers - 1].numNeurons);
		for (int i = 0; i < map.size(); i++) {
			map[i].maxValue = max;
			map[i].minValue = min;
		}
	}
	float reverseMap(int neur, float val) {
		return (val - map[neur].minValue) / (map[neur].maxValue - map[neur].minValue);
	}
	vector<u_int> casualVector(int in, int start = 0) {
		// crea un vettore di n ellementi successivi e li disordina
		// creazione di una tabella di accesso casuale per un secondo vettore
		vector<u_int> out(in);
		for (int i = 0; i < out.size(); i++) out[i] = i + start;
		random_shuffle(out.begin(), out.end());
		return out;
	}
	/*vector<T, A>*/
	template<typename T, typename A>
	void shackeVector(vector<T, A> const& vec) {
		//esegue il mescolamento degli elementi all'interno di un oggetto vector 
		random_shuffle(vec.begin(), vec.end());
		//return vec;
	}
	void refreshNeurIdx() {
		int idx = 0;
		for (int i = 0; i < nLayers; i++) {
			for (int j = 0; j < numNeurons(i); j++) {
				getTarget(i, j)->neurIdx = idx++;
			}
		}
	}
	void datasetOffset(float offset) {
		for (int i = 0; i < examples.size(); i++) {
			for (int j = 0; j < examples[i].Doutput.size(); j++) {
				examples[i].Doutput[j] += offset;
			}
		}
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////ACCESSO A VARIABILI PRIVATE///////////////////////////////////////////////////
	int numLayers() { return nLayers; }
	int numNeurons(int Layer) { return Layers[Layer].numNeurons; }
	int numCon(int Layer, int Neuron) { return Layers[Layer].Neurons[Neuron].numOutArcs; }
	int numInCon(int Layer, int Neuron) { return Layers[Layer].Neurons[Neuron].numInArcs; }
	int getConTargetLyr(int Layer, int Neuron, int Arc) { return (int)Layers[Layer].Neurons[Neuron].OutArcs[Arc].target->layer; }
	int getConTargetCol(int Layer, int Neuron, int Arc) { return (int)Layers[Layer].Neurons[Neuron].OutArcs[Arc].target->column; }
	float getWeight(int Layer, int Neuron, int Arc) { return Layers[Layer].Neurons[Neuron].OutArcs[Arc].weight; }
	float getDeltaWeight(int Layer, int Neuron, int Arc) { return Layers[Layer].Neurons[Neuron].OutArcs[Arc].oldDelta; }
	float getOutput(int Layer, int Neuron) { return Layers[Layer].Neurons[Neuron].output; }
	float getBPerr(int Layer, int Neuron) { return Layers[Layer].Neurons[Neuron].BPerr; }
	ptNeuron getTarget(int Layer, int Neuron) { return &(Layers[Layer].Neurons[Neuron]); }
	ptNeuron getConTarget(int Layer, int Neuron, int Conn) { return Layers[Layer].Neurons[Neuron].OutArcs[Conn].target; }
	int getOutConTargetID(ptNeuron base, ptNeuron target) {
		for (int i = 0; i < base->numOutArcs; i++) {
			if (base->OutArcs[i].target == target) {
				return i;
			}
		}
		cout << "connessione non trovata ERRORE!" << endl;
		return -1;
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////WINDOWS HIGH SPEED TIMING//////////////////////////////////////////////////////
	BOOL WINAPI QueryPerformanceCounter(_Out_ LARGE_INTEGER *lpPerformanceCount);
	BOOL WINAPI QueryPerformanceFrequency(_Out_ LARGE_INTEGER *lpFrequency);
	inline long long PerformanceCounter() noexcept
	{
		LARGE_INTEGER li;
		::QueryPerformanceCounter(&li);
		return li.QuadPart;
	}
	inline long long PerformanceFrequency() noexcept
	{
		LARGE_INTEGER li;
		::QueryPerformanceFrequency(&li);
		return li.QuadPart;
	}
	/* HOW TO USE:
	long long t0 = PerformanceCounter();
	//code to bench..
	long long t1 = PerformanceCounter();
	double elapsedMilliseconds = ((t1 - t0) * 1000.0) / PerformanceFrequency();
	*/
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
};

class MLP : public Network {
protected:
	//Hopfield *Supporter;

public:

	float NetPerformance = 0; // tempo di esecuzione medio in millisecondi
	float NetErrPercent = 0; //errore percentuale medio associato alla rete

	MLP(string filename) :Network(filename) {};

	////////////////////////////////////////////////FUNZIONI CREAZIONE RETE///////////////////////////////////////////////////////
	//CREAZIONE RETE QUADRATA
	void qubeNet(int Nlayers, int Ncolumns, int input, int output, bool c, float initValue = 0.01f) { //TODO inserire n� neuroni di input e output!!
		// testata e funzionante
		// nlayers - numero layer della rete
		// Ncolumns - numero neuroni per strato
		// c - se true inizializa casualmente i pesi altrimenti sono inizializati tutti a 1
		Layers.resize(Nlayers, Layer()); //= new vector<Layer>(stoi(segment)); // dichiarazione numero di layer
		nLayers = Nlayers;
		cout << "Declaring structure...." << endl;

		//per lo strato di input
		Layers[0].numNeurons = input;
		Layers[0].Neurons.resize(input, Neuron());
		nNeurons += input;
		for (int j = 0; j < input; j++) {
			Layers[0].Neurons[j].OutArcs.resize(Ncolumns, arc());
			Layers[0].Neurons[j].numOutArcs = Ncolumns;
			Layers[0].Neurons[j].layer = 0;
			Layers[0].Neurons[j].column = j;
			nArc += Ncolumns;
		}

		for (int i = 1; i < Nlayers - 1; i++) {
			// per gli strati intermedi
			Layers[i].numNeurons = Ncolumns;
			Layers[i].Neurons.resize(Ncolumns, Neuron());
			nNeurons += Ncolumns;
			for (int j = 0; j < Ncolumns; j++) {
				if (i < Nlayers - 2) {
					Layers[i].Neurons[j].OutArcs.resize(Ncolumns, arc());
					Layers[i].Neurons[j].numOutArcs = Ncolumns;
					nArc += Ncolumns;
				}
				else {
					Layers[i].Neurons[j].OutArcs.resize(output, arc());
					Layers[i].Neurons[j].numOutArcs = output;
					nArc += output;
				}
				Layers[i].Neurons[j].layer = i;
				Layers[i].Neurons[j].column = j;
			}

		}
		//per lo strato di output
		Layers[Nlayers - 1].numNeurons = output;
		Layers[Nlayers - 1].Neurons.resize(output, Neuron());
		nNeurons += output;
		for (int j = 0; j < output; j++) {
			Layers[Nlayers - 1].Neurons[j].layer = Nlayers - 1;
			Layers[Nlayers - 1].Neurons[j].column = j;
		}

		float initWeight = initValue;
		float Q = sqrt(3)*sqrt(2);
		int M = 10;
		cout << "initializing wheights...." << endl;

		for (int j = 0; j < input; j++) {
			for (int k = 0; k < Ncolumns; k++) {
				if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() - 80); initWeight = (rand() % M) / Q; } }
				Layers[0].Neurons[j].OutArcs[k].target = &(Layers[1].Neurons[k]);
				Layers[0].Neurons[j].OutArcs[k].weight = initWeight;
				Layers[1].Neurons[k].numInArcs++;
			}
		}

		for (int i = 1; i < Nlayers - 2; i++) {
			//cout << "Layer:" << i << endl;
			for (int j = 0; j < Ncolumns; j++) {
				for (int k = 0; k < Ncolumns; k++) {
					if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() + 50); initWeight = (rand() % M) / Q; } }
					Layers[i].Neurons[j].OutArcs[k].target = &(Layers[i + 1].Neurons[k]);
					Layers[i].Neurons[j].OutArcs[k].weight = initWeight;
					Layers[i + 1].Neurons[k].numInArcs++;
				}
			}
		}

		for (int j = 0; j < Ncolumns; j++) {
			for (int k = 0; k < output; k++) {
				if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() / 0.7556); initWeight = (rand() % M) / Q; } }
				Layers[Nlayers - 2].Neurons[j].OutArcs[k].target = &(Layers[Nlayers - 1].Neurons[k]);
				Layers[Nlayers - 2].Neurons[j].OutArcs[k].weight = initWeight;
				Layers[Nlayers - 1].Neurons[k].numInArcs++;
			}
		}
		refreshNeurIdx();
	}
	//CREAZIONE RETE QUADRATA COMPLETAMENTE CONNESSA
	void qubeNetFC(int Nlayers, int Ncolumns, int input, int output, bool c, float initValue = 0.01f) {
		//TODO inserire n� neuroni di input e output!!
		// testata e funzionante
		// nlayers - numero layer della rete
		// Ncolumns - numero neuroni per strato
		// c - se true inizializa casualmente i pesi altrimenti sono inizializati tutti a 1
		Layers.resize(Nlayers, Layer()); //= new vector<Layer>(stoi(segment)); // dichiarazione numero di layer
		nLayers = Nlayers;
		cout << "Declaring structure...." << endl;

		//per lo strato di input
		Layers[0].numNeurons = input;
		Layers[0].Neurons.resize(input, Neuron());
		nNeurons += input;
		int numA = (nLayers - 2)*Ncolumns + output;
		nArc += input * numA;
		for (int j = 0; j < input; j++) {
			Layers[0].Neurons[j].OutArcs.resize(numA, arc());
			Layers[0].Neurons[j].numOutArcs = numA;
			Layers[0].Neurons[j].layer = 0;
			Layers[0].Neurons[j].column = j;
		}

		for (int i = 1; i < Nlayers - 1; i++) {
			// per gli strati intermedi
			numA = (Nlayers - i - 2)*Ncolumns + output;
			Layers[i].numNeurons = Ncolumns;
			Layers[i].Neurons.resize(Ncolumns, Neuron());
			nNeurons += Ncolumns;
			for (int j = 0; j < Ncolumns; j++) {
				if (i < Nlayers - 2) {
					Layers[i].Neurons[j].OutArcs.resize(numA, arc());
					Layers[i].Neurons[j].numOutArcs = numA;
					nArc += numA;
				}
				else {
					Layers[i].Neurons[j].OutArcs.resize(output, arc());
					Layers[i].Neurons[j].numOutArcs = output;
					nArc += output;
				}
				Layers[i].Neurons[j].layer = i;
				Layers[i].Neurons[j].column = j;
			}

		}
		//per lo strato di output
		Layers[Nlayers - 1].numNeurons = output;
		Layers[Nlayers - 1].Neurons.resize(output, Neuron());
		nNeurons += output;
		for (int j = 0; j < output; j++) {
			Layers[Nlayers - 1].Neurons[j].layer = Nlayers - 1;
			Layers[Nlayers - 1].Neurons[j].column = j;
		}

		float initWeight = initValue;
		float Q = sqrt(3)*sqrt(2);
		int M = 10;
		int arcRef = 0;
		cout << "initializing weights...." << endl;

		// inizializzo pesi strato input
		for (int j = 0; j < input; j++) {
			for (int k = 0; k < (Nlayers - 2); k++) {
				for (int n = 0; n < Ncolumns; n++) {
					numA = k * Ncolumns + n;
					if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() - 80); initWeight = (rand() % M) / Q; } }
					Layers[0].Neurons[j].OutArcs[numA].target = &(Layers[k + 1].Neurons[n]);
					Layers[0].Neurons[j].OutArcs[numA].weight = initWeight;
					Layers[k + 1].Neurons[n].numInArcs++;
				}
			}
			for (int n = 0; n < output; n++) {
				numA = (Nlayers - 2) * Ncolumns + n;
				if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() - 80); initWeight = (rand() % M) / Q; } }
				Layers[0].Neurons[j].OutArcs[numA].target = &(Layers[Nlayers - 1].Neurons[n]);
				Layers[0].Neurons[j].OutArcs[numA].weight = initWeight;
				Layers[Nlayers - 1].Neurons[n].numInArcs++;
			}
		}
		//inizializzo pesi strati intermedi
		for (int i = 1; i < Nlayers - 2; i++) {
			//cout << "Layer:" << i << endl;
			for (int j = 0; j < Ncolumns; j++) {
				for (int k = i + 1; k < Nlayers - 1; k++) {
					for (int w = 0; w < Ncolumns; w++) {
						numA = (k - i - 1)*Ncolumns + w;
						if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() + 50); initWeight = (rand() % M) / Q; } }
						Layers[i].Neurons[j].OutArcs[numA].target = &(Layers[k].Neurons[w]);
						Layers[i].Neurons[j].OutArcs[numA].weight = initWeight;
						Layers[k].Neurons[w].numInArcs++;
					}
				}
				for (int k = 0; k < output; k++) {
					numA = (Nlayers - 2 - i)*Ncolumns + k;
					if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() + 50); initWeight = (rand() % M) / Q; } }
					Layers[i].Neurons[j].OutArcs[numA].target = &(Layers[Nlayers - 1].Neurons[k]);
					Layers[i].Neurons[j].OutArcs[numA].weight = initWeight;
					Layers[Nlayers - 1].Neurons[k].numInArcs++;
				}
			}
		}
		//inizializzo pesi strato output
		for (int j = 0; j < Ncolumns; j++) {
			for (int k = 0; k < output; k++) {
				if (c) { initWeight = 0; while (abs(initWeight) > 0.05) { srand(clock() / 0.7556); initWeight = (rand() % M) / Q; } }
				Layers[Nlayers - 2].Neurons[j].OutArcs[k].target = &(Layers[Nlayers - 1].Neurons[k]);
				Layers[Nlayers - 2].Neurons[j].OutArcs[k].weight = initWeight;
				Layers[Nlayers - 1].Neurons[k].numInArcs++;
			}
		}
		refreshNeurIdx();
	}
	//CREAZIONE RETE CUSTOM
	void customNet(int Nlayers, vector<int> Ncolumns, float conFill) {
		//TODO aggiungere la conta delle connessioni in nArc durante la dichiarazione
		if (Ncolumns.size() != Nlayers) { cout << "costumNet() FAILED! parameters error" << endl; return; } //errore
																											//dichiarazione layers
		Layers.resize(Nlayers);
		nLayers = Nlayers;
		//dichiarazione neuroni per layer
		for (int i = 0; i < Nlayers; i++) {
			Layers[i].Neurons.resize(Ncolumns[i]);
			Layers[i].numNeurons = Ncolumns[i];
			nNeurons += Ncolumns[i];
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				Layers[i].Neurons[j].layer = i;
				Layers[i].Neurons[j].column = j;
			}
		}
		//dichiarazione connessioni in uscita di ogni neurone
		for (int i = 0; i < nLayers - 1; i++) {
			cout << "Layer " << i << endl;
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				cout << "Neuron " << j << endl;
				int nFrontNeurons = 0; for (int g = i + 1; g < nLayers; g++) nFrontNeurons += Layers[g].numNeurons;
				Layers[i].Neurons[j].numOutArcs = conFill * nFrontNeurons;
				Layers[i].Neurons[j].OutArcs.resize(conFill* nFrontNeurons);

				vector<u_int> OutArcAcces = casualVector(Layers[i].Neurons[j].numOutArcs);
				vector<u_int> NetAcces = casualVector(nFrontNeurons);
				int x, y = 0;

				while (y < OutArcAcces.size()) {
					x = 0;
					for (int k = i + 1; k < nLayers; k++) {
						for (int w = 0; w < Layers[k].numNeurons; w++) {
							if (y > OutArcAcces.size() - 1) continue;
							if (NetAcces[y] == x) {
								Layers[i].Neurons[j].OutArcs[OutArcAcces[y]].target = &(Layers[k].Neurons[w]);
								Layers[i].Neurons[j].OutArcs[OutArcAcces[y]].weight = 0.01f;
								Layers[k].Neurons[w].numInArcs++;
								y++; cout << y << endl;
							}
							x++;
						}
					}
				}
			}
		}
		refreshNeurIdx();
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////STIMOLAZIONE RETE/////////////////////////////////////////////////////////////////
	//procedura di propagazione dell'informazione
	void inputNet(vector<float> &input, vector<float> &output) {
		if (input.size() != Layers[0].numNeurons) { cout << "il vettore in input deve avere dim " << Layers[0].numNeurons << endl; return; }
		if (output.size() != Layers[nLayers - 1].numNeurons) { cout << "il vettore in output deve avere dim " << Layers[nLayers - 1].numNeurons << endl; return; }

		// carico il vettore di input nella rete
		for (int i = 0; i < Layers[0].numNeurons; i++) {
			Layers[0].Neurons[i].output = input[i];
		}
		//sigLayer(0);

		// propagazione dello stimolo
		for (int i = 0; i < nLayers - 1; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				for (int k = 0; k < Layers[i].Neurons[j].numOutArcs; k++) {
					Layers[i].Neurons[j].OutArcs[k].target->output += Layers[i].Neurons[j].output*Layers[i].Neurons[j].OutArcs[k].weight;
				}
			}
			bayesLayer(i + 1);
			sigLayer(i + 1);
		}

		float delta;

		if (map.size() == 0) { cout << "Errore ... la mappatura degli output non � stata settata!!"; return; }

		for (int i = 0; i < Layers[nLayers - 1].numNeurons; i++) {
			delta = map[i].maxValue - map[i].minValue;
			output[i] = ((Layers[nLayers - 1].Neurons[i].output)*delta) + map[i].minValue;
			Layers[nLayers - 1].Neurons[i].output = output[i];
		}
	}
	//esegue una propagazione dell'informazione salvando lo storico di propagazione degli input
	void inputNetProfiler(vector<float> &input, vector<float> &output) {
		if (input.size() != Layers[0].numNeurons) { cout << "il vettore in input deve avere dim " << Layers[0].numNeurons << endl; return; }
		if (output.size() != Layers[nLayers - 1].numNeurons) { cout << "il vettore in output deve avere dim " << Layers[nLayers - 1].numNeurons << endl; return; }

		// carico il vettore di input nella rete
		for (int i = 0; i < Layers[0].numNeurons; i++) {
			Layers[0].Neurons[i].output = input[i];
			//Layers[0].Neurons[i].absOutSum = abs(input[i]);
		}
		//sigLayer(0);
		// propagazione dello stimolo
		for (int i = 0; i < nLayers - 1; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				for (int k = 0; k < Layers[i].Neurons[j].numOutArcs; k++) {
					Layers[i].Neurons[j].OutArcs[k].target->output += Layers[i].Neurons[j].output*Layers[i].Neurons[j].OutArcs[k].weight;
					Layers[i].Neurons[j].OutArcs[k].target->absOutSum += abs(Layers[i].Neurons[j].output*Layers[i].Neurons[j].OutArcs[k].weight);
				}
			}

			bayesLayer(i + 1, true);
			sigLayer(i + 1);
		}

		resetVectorsProfiler(true, false);// resetto tutti i vettori
		//inizializzo la propagazione dal primo layer
		int IDinf = 0;
		for (int i = 0; i < Layers[1].numNeurons; i++) { // scorro i neuroni del primo strato nascosto
			for (int j = 0; j < Layers[0].numNeurons; j++) { // scorro i neuroni dello strato di input 
				for (int k = 0; k < numCon(0, j); k++) { // scorro le connessioni del neurone dello strato di input
					if (&Layers[1].Neurons[i] == Layers[0].Neurons[j].OutArcs[k].target) {
						Layers[1].Neurons[i].influenceInput[IDinf] = (abs(getTarget(0, j)->output*getTarget(0, j)->OutArcs[k].weight) / (getTarget(0, j)->OutArcs[k].target->absOutSum));
						IDinf++;
					}
				}
			}
			IDinf = 0;
		}
		//inizzializzo la propagazione negli altri layer
		for (int t = 1; t < nLayers - 1; t++) { // togliere il meno uno
			IDinf = 0;
			for (int i = 0; i < Layers[t + 1].numNeurons; i++) {
				for (int j = 0; j < t + 1; j++) {
					for (int w = 0; w < Layers[j].numNeurons; w++) {
						for (int k = 0; k < numCon(j, w); k++) {
							if (&Layers[t + 1].Neurons[i] == Layers[j].Neurons[w].OutArcs[k].target) {
								Layers[t + 1].Neurons[i].influenceInput[IDinf] = (abs(getTarget(j, w)->output*getTarget(j, w)->OutArcs[k].weight) / (getTarget(j, w)->OutArcs[k].target->absOutSum));
								IDinf++;
							}
						}
					}
				}
				IDinf = 0;
			}
		}


		float delta;

		if (map.size() == 0) {
			cout << "Errore ... la mappatura degli output non � stata settata!!" << endl;
			return;
		}

		for (int i = 0; i < Layers[nLayers - 1].numNeurons; i++) {
			delta = map[i].maxValue - map[i].minValue;
			output[i] = ((Layers[nLayers - 1].Neurons[i].output)*delta) + map[i].minValue;
			Layers[nLayers - 1].Neurons[i].output = output[i];
		}
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////FUNZIONI DI ADDESTRAMENTO MLP////////////////////////////////////////////////////////
	//Algoritmo di addestramento Back-propagation
	void BP(int iter, float eps, float beta, float ErrPercent) {
		float err = 0; // errore del singolo neurone
		float Err = 0; // errore quadratio del singolo esempio
		float Gerr = 0; // errore quadratico dell'intero dataset
		float Perr = 0;
		float delta = 0;
		long long t0 = 0;
		long long t1 = 0;
		long long inT1 = 0;
		double elapsedMilliseconds = 0;
		double executionTime = 0;
		double inputTime = 0;
		int x = 0;
		vector<float> Out(numNeurons(nLayers - 1)); // vettore supporto output rete
		vector<float> GerrStory;

		for (int t = 0; t < iter; t++) {
			Gerr = 0;
			Perr = 0;
			for (int e = 0; e < examples.size(); e++) {
				t0 = PerformanceCounter();
				// eseguo l'esempio
				inputNet(examples[e].input, Out);

				t1 = PerformanceCounter();
				elapsedMilliseconds = ((t1 - t0) * 1000.0) / PerformanceFrequency();
				inputTime += elapsedMilliseconds;

				// clacolo il delta delle variazioni dei pesi dello strato di output
				Err = 0;
				for (int r = 0; r < Out.size(); r++) {
					delta = map[r].maxValue - map[r].minValue;
					err = Out[r] - examples[e].Doutput[r];
					Perr += abs(err / examples[e].Doutput[r]) * 100; // errore percentuale
					Err += pow(err, 2);
					if (x == e && t > -1) {
						cout << "es." << x << " Y" << r << " = " << Out[r] << "   " << "D" << r << " = " << examples[e].Doutput[r] << "  err: " << abs(Out[r] - examples[e].Doutput[r]) << endl;
						if (r == Out.size() - 1) { x--; WeightsStamp("fc"); /*Sleep(5000);*/ }
						if (x < 0) x = examples.size() - 1;
					}
					//mErr = ((err- map[r].minValue)/ delta) ;
					// calcolo il delta della variazione dei pesi
					Layers[nLayers - 1].Neurons[r].BPerr = reverseMap(r, err) * (reverseMap(r, Out[r]) *(1 - reverseMap(r, Out[r])));
					//applico le correzioni ai bayes
					Layers[nLayers - 1].Neurons[r].oldBayesDelta = (-eps * (Layers[nLayers - 1].Neurons[r].BPerr)) + beta * Layers[nLayers - 1].Neurons[r].oldBayesDelta;
					Layers[nLayers - 1].Neurons[r].bayes += Layers[nLayers - 1].Neurons[r].oldBayesDelta;
				}
				//applico le correzioni ai pesi dello strato di output
				for (int i = 0; i < numNeurons(nLayers - 2); i++) {
					for (int j = 0; j < numCon(nLayers - 2, i); j++) {

						Layers[nLayers - 2].Neurons[i].OutArcs[j].oldDelta = (-eps * (Layers[nLayers - 2].Neurons[i].OutArcs[j].target->BPerr)*Layers[nLayers - 2].Neurons[i].output) + beta * Layers[nLayers - 2].Neurons[i].OutArcs[j].oldDelta;
						Layers[nLayers - 2].Neurons[i].OutArcs[j].weight += Layers[nLayers - 2].Neurons[i].OutArcs[j].oldDelta;
					}
				}
				// rieseguo la procedura per tutti gli altri strati
				for (int i = nLayers - 2; i > 0; i--) { // dal penultimo strato al secondo
														// clacolo il delta delle variazioni dei pesi dello strato i-1
					for (int j = 0; j < numNeurons(i); j++) {
						err = 0;
						for (int k = 0; k < numCon(i, j); k++) err += Layers[i].Neurons[j].OutArcs[k].target->BPerr * Layers[i].Neurons[j].OutArcs[k].weight;
						Layers[i].Neurons[j].BPerr = DsigOut(i, j)*err;
						//applico le correzioni ai bayes
						Layers[i].Neurons[j].oldBayesDelta = (-eps * (Layers[i].Neurons[j].BPerr)) + beta * Layers[i].Neurons[j].oldBayesDelta;
						Layers[i].Neurons[j].bayes += Layers[i].Neurons[j].oldBayesDelta;

					}
					// applico le correzioni ai pesi dello strato i-1
					for (int j = 0; j < numNeurons(i - 1); j++) {
						for (int k = 0; k < numCon(i - 1, j); k++) {
							Layers[i - 1].Neurons[j].OutArcs[k].oldDelta = (-eps * (Layers[i - 1].Neurons[j].OutArcs[k].target->BPerr)*Layers[i - 1].Neurons[j].output) + beta * Layers[i - 1].Neurons[j].OutArcs[k].oldDelta;
							Layers[i - 1].Neurons[j].OutArcs[k].weight += Layers[i - 1].Neurons[j].OutArcs[k].oldDelta;
						}
					}
				}
				Gerr += Err;
				resetPotential(); // azzero i potenziali di output della rete
				t1 = PerformanceCounter();
				elapsedMilliseconds = ((t1 - t0) * 1000.0) / PerformanceFrequency();
				executionTime += elapsedMilliseconds;
			}

			Gerr /= 2;
			Perr /= examples.size()*Out.size();
			executionTime /= examples.size();
			inputTime /= examples.size();
			NetPerformance = inputTime;
			NetErrPercent = Perr;
			cout << "Iterazione  " << t << " errore quadratico: " << Gerr << "   errore percentuale medio:  " << Perr << "  %err  exemple time: " << executionTime << " ms  inputTime: " << inputTime << " ms" << endl;

			//if (t > 100 && t < 200)Sleep(500);
			if (Perr < ErrPercent) {
				cout << "Percentuale di errore obbiettivo raggiunta!" << endl;
				return;
			}

		}
	}
	//esecuzione del backpropagation per un solo esempio
	void oneBP(float eps, float beta, example e) {
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////ALTRE FUNZIONI MLP///////////////////////////////////////////////////////////

	void initVectorsProfiler() {
		//inizializza i vettori di benchmark presenti nei neuroni
		//inizializzazione vettori del primo strato
		for (int j = 0; j < Layers[0].numNeurons; j++) {
			Layers[0].Neurons[j].influenceOutput.resize(Layers[0].Neurons[j].numOutArcs);
			fill(Layers[0].Neurons[j].influenceOutput.begin(), Layers[0].Neurons[j].influenceOutput.end(), 0);
		}
		for (int i = 1; i < nLayers - 1; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				Layers[i].Neurons[j].influenceInput.resize(Layers[i].Neurons[j].numInArcs);
				Layers[i].Neurons[j].influenceOutput.resize(Layers[i].Neurons[j].numOutArcs);
				// resetto tutti i vettori  zero
				fill(Layers[i].Neurons[j].influenceInput.begin(), Layers[i].Neurons[j].influenceInput.end(), 0);
				fill(Layers[i].Neurons[j].influenceOutput.begin(), Layers[i].Neurons[j].influenceOutput.end(), 0);
			}
		}
		//inizializzazione dei vettori dell'ultimo strato
		for (int j = 0; j < Layers[nLayers - 1].numNeurons; j++) {
			Layers[nLayers - 1].Neurons[j].influenceInput.resize(Layers[nLayers - 1].Neurons[j].numInArcs);
			fill(Layers[nLayers - 1].Neurons[j].influenceInput.begin(), Layers[nLayers - 1].Neurons[j].influenceInput.end(), 0);
		}
	}
	void resetVectorsProfiler(bool inInfl, bool outInfl) { //resetta a zero tutti i vettori di profilazione esclusi layer input e output
		for (int i = 0; i < nLayers; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				if (i > 0 && inInfl == true)fill(Layers[i].Neurons[j].influenceInput.begin(), Layers[i].Neurons[j].influenceInput.end(), 0);
				if (i < nLayers - 1 && outInfl == true)fill(Layers[i].Neurons[j].influenceOutput.begin(), Layers[i].Neurons[j].influenceOutput.end(), 0);
			}
		}
	}
	//stampa a schermo l'influenza degli input per ogni uscita
	void stampInputInfluences(bool all = false) {
		cout << endl;
		if (all == false) {
			for (int i = 0; i < Layers[nLayers - 1].numNeurons; i++) {
				cout << "Output (" << i << "): ";
				for (int j = 0; j < Layers[nLayers - 1].Neurons[i].influenceInput.size(); j++) {
					cout << "infl n in(" << j << "): " << Layers[nLayers - 1].Neurons[i].influenceInput[j] * 100 << "% || ";
				}
				cout << endl;
			}
		}
		else {
			int l = 0;
			int c = 0;
			for (int i = 0; i < nLayers; i++) {
				for (int j = 0; j < Layers[i].numNeurons; j++) {
					cout << "neuron (" << i << ", " << j << ") input influence: ";
					for (int k = 0; k < Layers[i].Neurons[j].influenceInput.size(); k++) {
						l = basePosfromtarget(getTarget(i, j), k)->layer;
						c = basePosfromtarget(getTarget(i, j), k)->column;
						cout << " (" << l << ", " << c << "): " << Layers[i].Neurons[j].influenceInput[k] * 100 << "% || ";// non mostra la correlazione con il neurone ma solo in numero di arco posizionale per il neurone
					}
					cout << endl;
				}

			}
		}
	}
	//stampa a schermo l'influenza degli errori degli output per ogni ingresso
	void stampOutputErrorPropagation(bool all = false) {
		cout << endl;
		if (all == false) {
			for (int i = 0; i < Layers[0].numNeurons; i++) {
				cout << "input (" << i << ") BPerrore : ";
				for (int j = 0; j < Layers[0].Neurons[i].influenceOutput.size(); j++) {
					cout << " (" << j << "): " << Layers[0].Neurons[i].influenceOutput[j] * 100 << "% || ";
				}
				cout << endl;
			}
		}
		else {
			for (int i = 0; i < nLayers; i++) {
				for (int j = 0; j < Layers[i].numNeurons; j++) {
					cout << "neuron (" << i << ", " << j << "): ";
					for (int k = 0; k < Layers[i].Neurons[j].influenceOutput.size(); k++) {
						cout << "BPerr n out(" << k << "): " << Layers[i].Neurons[j].influenceOutput[k] * 100 << "% || "; // non mostra la correlazione con il neurone ma solo in numero di arco posizionale per il neurone
					}
					cout << endl;
				}

			}
		}
	}
	// dal riferimento al nodo target e dall'indice del vettore di influenza restituisce il puntatore al neurone base
	ptNeuron basePosfromtarget(ptNeuron target, int k) {
		int k2 = -1;
		for (int i = 0; i < target->layer; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				for (int w = 0; w < numCon(i, j); w++) {
					if (target == getConTarget(i, j, w)) {
						k2++;
						if (k2 == k) return getTarget(i, j);
					}
				}
			}
		}
		return nullptr;
	}
	//dato un neurone e l'indice di un suo arco restituisce l'indice di quella connessione all'interno del vettore influenceInput all'interno del neurone target
	int idBaseConReftoTargetInfl(ptNeuron base, int arc) {
		ptNeuron target = base->OutArcs[arc].target;
		int k2 = -1;
		for (int i = 0; i < target->layer; i++) {
			for (int j = 0; j < numNeurons(i); j++) {
				for (int k = 0; k < numCon(i, j); k++) {
					if (target == getConTarget(i, j, k)) {
						k2++;
						if (base == getTarget(i, j)) {
							return k2;
						}
					}
				}
			}
		}
		return -1;
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

};

class Hopfield : public Network {

public:

	MLP* mlp; // puntatore alla rete da supportare
	vector<interArc> binds; // connessioni tra rete mlp e Hopfiled

	Hopfield(string filename, MLP *target = NULL) :Network(filename) {
		if (target != NULL) {
			mlp = target;
			supportNet();
		}
	}

	////////////////////////////////////////////////FUNZIONI CREAZIONE RETE Hopfield///////////////////////////////////////////////
	//Rete ad anello //RICORDA i pesi sono settati automaticamente a 1 TODO implementa controllo da parametro
	void toroidNet(int Nlayers, vector<int> Ncolumns, float conFill) {
		//sviluppo simile alle reti quadrate FC ma con connessioni anche tra i neuroni di uno stesso strato
		if (Ncolumns.size() != Nlayers) { cout << "ringNet() FAILED! parameters error" << endl; return; } //errore
																										  //dichiarazione layers
		Layers.resize(Nlayers);
		nLayers = Nlayers;
		//dichiarazione neuroni per layer
		for (int i = 0; i < Nlayers; i++) {
			Layers[i].Neurons.resize(Ncolumns[i]);
			Layers[i].numNeurons = Ncolumns[i];
			nNeurons += Ncolumns[i];
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				Layers[i].Neurons[j].layer = i;
				Layers[i].Neurons[j].column = j;
			}
		}
		//dichiarazione connessioni in uscita di ogni neurone
		for (int i = 0; i < nLayers; i++) {
			//cout << "Layer " << i << endl;
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				//cout << "Neuron " << j << endl;
				int nFrontNeurons = 0; for (int g = i; g < nLayers; g++) nFrontNeurons += Layers[g].numNeurons;
				Layers[i].Neurons[j].numOutArcs = conFill * nFrontNeurons;
				Layers[i].Neurons[j].OutArcs.resize(conFill* nFrontNeurons);

				vector<u_int> OutArcAcces = casualVector(Layers[i].Neurons[j].numOutArcs);
				vector<u_int> NetAcces = casualVector(nFrontNeurons);

				int x, y = 0;

				while (y < OutArcAcces.size()) {
					x = 0;
					for (int k = i; k < nLayers; k++) {
						for (int w = 0; w < Layers[k].numNeurons; w++) {
							if (y > OutArcAcces.size() - 1) continue;
							if (NetAcces[y] == x) {
								Layers[i].Neurons[j].OutArcs[OutArcAcces[y]].target = &(Layers[k].Neurons[w]);
								Layers[i].Neurons[j].OutArcs[OutArcAcces[y]].weight = 1.0f;
								Layers[k].Neurons[w].numInArcs++;
								y++; cout << y << endl;
							}
							x++;
						}
					}
				}
				//TOPPA elimino le connessioni dei neuroni con se stessi
				for (int k = 0; k < Layers[i].Neurons[j].numOutArcs; k++) {
					if (Layers[i].Neurons[j].OutArcs[k].target->layer == i) {
						if (Layers[i].Neurons[j].OutArcs[k].target->column == j) {
							deleteArc(i, j, i, j);
						}
					}
				}
			}
		}
	}
	//Genera una rete di supporto all'apprendimento su misura per una mlp 
	void supportNet() {
		//creo la rete di supporto
		nLayers = 1;
		nNeurons = mlp->nNeurons;
		nArc = mlp->nArc;
		Layers.resize(1);
		Layers[0].Neurons.resize(nNeurons);
		Layers[0].numNeurons = nNeurons;
		//collego la rete di supporto alla mlp
		//e copio le connessioni dalla mlp sulla hopfield
		binds.resize(mlp->nNeurons);
		int bindPos = 0;
		for (int i = 0; i < mlp->nLayers; i++) {
			for (int j = 0; j < mlp->numNeurons(i); j++) {
				binds[bindPos].base = getTarget(0, bindPos);
				binds[bindPos].base->layer = 0;
				binds[bindPos].base->column = bindPos;
				binds[bindPos].target = mlp->getTarget(i, j);
				binds[bindPos].base->numOutArcs = binds[bindPos].target->numOutArcs;
				binds[bindPos].base->numInArcs = binds[bindPos].target->numInArcs;
				binds[bindPos].base->OutArcs.resize(binds[bindPos].target->numOutArcs);
				bindPos++;
			}
		}

		for (int i = 0; i < binds.size(); i++) {
			for (int k = 0; k < binds[i].target->numOutArcs; k++) {
				int TargetPos = searchFromTarget(binds[i].target->OutArcs[k].target);
				if (TargetPos != -1) {
					binds[i].base->OutArcs[k].target = binds[TargetPos].base;
					binds[i].base->OutArcs[k].weight = 1.0f;
				}
			}
		}
	}
	//Genera una rete con connessioni completamente caotiche
	void caoticNet() {} // TODO da sviluppare
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////FUNZIONI DI ADDESTRAMENTO//////////////////////////////////////////////////////
	//la funzione considera i parametri della rete mlp associta per addestrare la memoria associativa 
	void trainSupportNet(float Wi, float We, int g) {
		float infl = 0, BPerr = 0;
		float Dinfl = 0, DbpErr = 0;
		for (int i = 0; i < mlp->nLayers; i++) {
			for (int j = 0; j < mlp->numNeurons(i); j++) {
				for (int k = 0; k < mlp->numCon(i, j); k++) {
					BPerr = mlp->getTarget(i, j)->influenceOutput[k];
					infl = mlp->getTarget(i, j)->OutArcs[k].target->influenceInput[mlp->idBaseConReftoTargetInfl(mlp->getTarget(i, j), k)];
					Dinfl = DeltaHW(Wi, infl, g);
					DbpErr = DeltaHW(We, BPerr, g);
					binds[searchFromTarget(mlp->getTarget(i, j))].base->OutArcs[k].weight += Dinfl + DbpErr;
				}
			}
		}
	}
	//funzione che taglia gli archi con il peso sotto una certa soglia 
	void cutMadArc(int maxCut, int alfa) {
		//!!!la procedura supporta soltanto reti ad anello su un solo layer!!!
		//maxCut rappresenta il numero massimo di rami da tagliare in questa esecuzione
		//alfa rappresenta il coefficiente di soglia percentuale sotto il quale il ramo puo essere tagliato (la soglia � una percentuale)
		float Wsum = 0;
		u_int Wn = 0;
		float Wmax = std::numeric_limits<float>::min();
		float Wmin = std::numeric_limits<float>::max();
		for (int i = 0; i < numNeurons(0); i++) {
			for (int j = 0; j < numCon(0, i); j++) {
				Wsum += getTarget(0, i)->OutArcs[j].weight;
				if (getTarget(0, i)->OutArcs[j].weight > Wmax) { Wmax = getTarget(0, i)->OutArcs[j].weight; }
				if (getTarget(0, i)->OutArcs[j].weight < Wmin) { Wmax = getTarget(0, i)->OutArcs[j].weight; }
				Wn++;
			}
		}
		nArc = Wn;
		mlp->nArc = nArc;
		float Wmean = Wsum / Wn;
		float treashold = ((Wmax - Wmin)*alfa) + Wmin; // alfa = 0 -> Treashold = Wmin , alfa = 1 -> Treashold = Wmax
		int Tcol = 0, TmlpL = 0, TmlpC = 0, mlpL = 0, mlpC = 0;
		vector <u_int> acs = casualVector(nNeurons - mlp->numNeurons(mlp->nLayers - 1));

		for (int i = 0; i < acs.size(); i++) {
			for (int j = 0; j < getTarget(0, acs[i])->numOutArcs; j++) {
				if (getTarget(0, acs[i])->OutArcs[j].weight < treashold) { // condizione sufficiente all'eliminazione

					Tcol = getTarget(0, acs[i])->OutArcs[j].target->column; //colonna del neurone Hopfield da eliminare
					TmlpL = binds[serchFromBase(getTarget(0, acs[i])->OutArcs[j].target)].target->layer; // numLayer del neurone mlp da eliminare
					TmlpC = binds[serchFromBase(getTarget(0, acs[i])->OutArcs[j].target)].target->column; // numCol del neurone mlp da eliminare
					mlpL = binds[serchFromBase(getTarget(0, acs[i]))].target->layer; // numLayer del neurone mlp contenente la connessione
					mlpC = binds[serchFromBase(getTarget(0, acs[i]))].target->column; // numCol del neurone mlp contenente la connessione

					// procedura di eliminazione arco
					if (getTarget(0, acs[i])->OutArcs[j].target->numInArcs == 1) { // se il neurone obbiettivo ha solo l'arco da eliminare elimino anche il corrispondente elemento bind
						//binds.erase(binds.begin() + serchFromBase(getTarget(0, acs[i])->OutArcs[j].target)); //cerco e elimino l'elemento bind dei due neuroni
						//deleteArc(0, acs[i], 0, Tcol); // elimino l'arco nella rete Hopfield
						//mlp->deleteArc(mlpL, mlpC, TmlpL, TmlpC); //elimino l'arco nella rete mlp

						//TODO utilizza funzioni apposite per l'eliminazione dei neuroni delle due reti

						nArc++;
					}
					else { //altrimenti elimino semplicemente l'arco nelle due reti
						deleteArc(0, acs[i], 0, Tcol); // elimino l'arco nella rete Hopfield
						mlp->deleteArc(mlpL, mlpC, TmlpL, TmlpC); //elimino l'arco nella rete mlp
					}
					nArc--;
					mlp->nArc = nArc;
					maxCut--;
					if (maxCut < 1) return;
				}
			}

		}
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////FUNZIONI MODIFICA RETE///////////////////////////////////////////////////////////
	void HdeleteArc(int Nlayer, int Ncolumn, int targetLayer, int targetColumn) {
		for (int i = 0; i < Layers[Nlayer].Neurons[Ncolumn].numOutArcs; i++) {
			if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->layer == targetLayer && Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->column == targetColumn) {

				//modifico i parametri della rete relazionati a tale arco
				Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->numInArcs--;
				if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->influenceInput.size() > 0)Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->influenceInput.pop_back();
				if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->influenceOutput.size() > 0)Layers[Nlayer].Neurons[Ncolumn].influenceOutput.pop_back();

				if (Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->numInArcs == 0 && Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->layer != 0) {
					deleteNeuron(Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->layer, Layers[Nlayer].Neurons[Ncolumn].OutArcs[i].target->column);
				}
				else {
					Layers[Nlayer].Neurons[Ncolumn].OutArcs.erase(Layers[Nlayer].Neurons[Ncolumn].OutArcs.begin() + i);
					Layers[Nlayer].Neurons[Ncolumn].numOutArcs--;
				}

				return;
			}
		}
		cout << "errore: arco (" << Nlayer << "-" << Ncolumn << ") -> (" << targetLayer << "-" << targetColumn << ") non trovato, eliinazione fallita" << endl;
	}
	void HdeleteArcByRef(int Nlayer, int Ncolumn, arc *targetArc) { //elimina un arco con gli indici della base e il puntatore alla connsessione

																   //modifico i parametri della rete relazionati a tale arco
		targetArc->target->numInArcs--;
		if (targetArc->target->influenceInput.size() > 0)targetArc->target->influenceInput.pop_back();
		if (Layers[Nlayer].Neurons[Ncolumn].influenceOutput.size() > 0)Layers[Nlayer].Neurons[Ncolumn].influenceOutput.pop_back();

		if (targetArc->target->numInArcs == 0) {
			deleteNeuron(targetArc->target->layer, targetArc->target->column);
		}
		else {
			Layers[Nlayer].Neurons[Ncolumn].OutArcs.erase(Layers[Nlayer].Neurons[Ncolumn].OutArcs.begin() + getOutConTargetID(getTarget(Nlayer, Ncolumn), targetArc->target));
			Layers[Nlayer].Neurons[Ncolumn].numOutArcs--;
		}

	}
	void HdeleteNeuron(int Nlayer, int Ncolumn) {
		// elimino l'arco dai neuroni che puntano al nodo da eliminare e
		// decremento il numero di archi in output hai neuroni con come target il neurone da eliminare
		for (int i = 0; i < Nlayer; i++) {
			for (int j = 0; j < Layers[i].numNeurons; j++) {
				for (int k = 0; k < Layers[i].Neurons[j].numOutArcs; k++) {
					if (Layers[i].Neurons[j].OutArcs[k].target->layer == Nlayer && Layers[i].Neurons[j].OutArcs[k].target->column == Ncolumn) {
						deleteArcByRef(i, j, &Layers[i].Neurons[j].OutArcs[k]);
					}
				}
			}
		}
		// eliminio gli archi del neurone da eliminare
		for (int i = 0; i < Layers[Nlayer].Neurons[Ncolumn].numOutArcs; i++) {
			deleteArcByRef(Nlayer, Ncolumn, &Layers[Nlayer].Neurons[Ncolumn].OutArcs[i]);
		}

		// decremento l'indice colonna dei neuroni successivi nello stesso layer
		for (int i = Ncolumn + 1; i < Layers[Nlayer].numNeurons; i++) Layers[Nlayer].Neurons[i].column--;

		// elimino il neurone
		Layers[Nlayer].Neurons.erase(Layers[Nlayer].Neurons.begin() + Ncolumn);
		Layers[Nlayer].numNeurons--;
		nNeurons--;
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////ALTRE FUNZIONI/////////////////////////////////////////////////////////////////
	int serchFromBase(ptNeuron base) {
		for (int i = 0; i < binds.size(); i++) {
			if (base == binds[i].base) {
				return i;
			}
		}
		return -1;
	}
	int searchFromTarget(ptNeuron target) {
		for (int i = 0; i < binds.size(); i++) {
			if (target == binds[i].target) {
				return i;
			}
		}
		return -1;
	}
	//funzione per il condizionamento dei valori dei vettori di retropropagazione
	float DeltaHW(float W, float x, float g) {
		return W * powf((2 * x - 1), (1 + 2 * g));
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
};

class StructuralLearning {
private:
	MLP * mlp; // rete mlp
	Hopfield* hpd; // rete Hopfield

public:

	StructuralLearning(MLP* ptMlp, Hopfield* ptHpd) { //costruttore
		mlp = ptMlp;
		hpd = ptHpd;
		mlp->initVectorsProfiler();
	}

	void StructuralBP(int iter, float eps, float beta, float ErrPercent, float Wi, float We, int g, int cutStart, float alfa, float SmaxErr, float maxWloss) {
		int nWt0 = mlp->nArc; // numero dei pesi al tempo t0
		int nWtn = 0; // numero dei pesi rimasti all'i-esima iterazzione
		float WdelP = 0; // percentuale dei pesi eliminati dall'inizio dell'addestramento
		float err = 0; // errore del singolo neurone
		float Err = 0; // errore quadratio del singolo esempio
		float Gerr = 0; // errore quadratico dell'intero dataset
		float Perr = 0;
		float delta = 0;
		long long t0 = 0;
		long long t1 = 0;
		long long inT1 = 0;
		double elapsedMilliseconds = 0;
		double executionTime = 0;
		double inputTime = 0;
		int x = 0;
		vector<float> Out(mlp->numNeurons(mlp->nLayers - 1)); // vettore supporto output rete
		vector<float> GerrStory;

		for (int t = 0; t < iter; t++) {
			Gerr = 0;
			Perr = 0;
			for (int e = 0; e < mlp->examples.size(); e++) {
				t0 = mlp->PerformanceCounter();
				// eseguo l'esempio
				mlp->inputNetProfiler(mlp->examples[e].input, Out);

				t1 = mlp->PerformanceCounter();
				elapsedMilliseconds = ((t1 - t0) * 1000.0) / mlp->PerformanceFrequency();
				inputTime += elapsedMilliseconds;

				// clacolo il delta delle variazioni dei pesi dello strato di output
				Err = 0;
				for (int r = 0; r < Out.size(); r++) {
					delta = mlp->map[r].maxValue - mlp->map[r].minValue;
					err = Out[r] - mlp->examples[e].Doutput[r];
					Perr += abs(err / mlp->examples[e].Doutput[r]) * 100; // errore percentuale
					Err += pow(err, 2);


					if (x == e && t > -1) {


						cout << endl << "es." << x << " Y" << r << " = " << Out[r] << "   " << "D" << r << " = " << mlp->examples[e].Doutput[r] << "  err: " << abs(Out[r] - mlp->examples[e].Doutput[r]) << endl;

						if (r == Out.size() - 1) {
							x--;
							mlp->WeightsStamp("fc");
							/*mlp->stampInputInfluences(true);
							mlp->stampOutputErrorPropagation(true);*/ //erroi retropropagati nell'iterazione precedente
							/*Sleep(5000);*/
						}
						if (x < 0) x = mlp->examples.size() - 1;
					}

					mlp->resetAbsSumDelta();

					//mErr = ((err- map[r].minValue)/ delta) ;
					// calcolo il delta della variazione dei pesi
					mlp->Layers[mlp->nLayers - 1].Neurons[r].BPerr = mlp->reverseMap(r, err) * (mlp->reverseMap(r, Out[r]) *(1 - mlp->reverseMap(r, Out[r])));
					//applico le correzioni ai bayes
					mlp->Layers[mlp->nLayers - 1].Neurons[r].oldBayesDelta = (-eps * (mlp->Layers[mlp->nLayers - 1].Neurons[r].BPerr)) + beta * mlp->Layers[mlp->nLayers - 1].Neurons[r].oldBayesDelta;
					mlp->Layers[mlp->nLayers - 1].Neurons[r].bayes += mlp->Layers[mlp->nLayers - 1].Neurons[r].oldBayesDelta;

				}
				//applico le correzioni ai pesi dello strato di output
				for (int i = 0; i < mlp->numNeurons(mlp->nLayers - 2); i++) {
					for (int j = 0; j < mlp->numCon(mlp->nLayers - 2, i); j++) {

						mlp->Layers[mlp->nLayers - 2].Neurons[i].OutArcs[j].oldDelta = (-eps * (mlp->Layers[mlp->nLayers - 2].Neurons[i].OutArcs[j].target->BPerr) * mlp->Layers[mlp->nLayers - 2].Neurons[i].output) + beta * mlp->Layers[mlp->nLayers - 2].Neurons[i].OutArcs[j].oldDelta;
						mlp->Layers[mlp->nLayers - 2].Neurons[i].OutArcs[j].weight += mlp->Layers[mlp->nLayers - 2].Neurons[i].OutArcs[j].oldDelta;
						mlp->Layers[mlp->nLayers - 2].Neurons[i].absDeltaSum += abs(mlp->Layers[mlp->nLayers - 2].Neurons[i].OutArcs[j].oldDelta);

					}
				}
				// rieseguo la procedura per tutti gli altri strati
				for (int i = mlp->nLayers - 2; i > 0; i--) { // dal penultimo strato al secondo
															 // clacolo il delta delle variazioni dei pesi dello strato i-1
					for (int j = 0; j < mlp->numNeurons(i); j++) {
						err = 0;
						for (int k = 0; k < mlp->numCon(i, j); k++) err += mlp->Layers[i].Neurons[j].OutArcs[k].target->BPerr * mlp->Layers[i].Neurons[j].OutArcs[k].weight;
						mlp->Layers[i].Neurons[j].BPerr = mlp->DsigOut(i, j)*err;
						//applico le correzioni ai bayes
						mlp->Layers[i].Neurons[j].oldBayesDelta = (-eps * (mlp->Layers[i].Neurons[j].BPerr)) + beta * mlp->Layers[i].Neurons[j].oldBayesDelta;
						mlp->Layers[i].Neurons[j].bayes += mlp->Layers[i].Neurons[j].oldBayesDelta;

					}
					// applico le correzioni ai pesi dello strato i-1
					for (int j = 0; j < mlp->numNeurons(i - 1); j++) {
						for (int k = 0; k < mlp->numCon(i - 1, j); k++) {

							mlp->Layers[i - 1].Neurons[j].OutArcs[k].oldDelta = (-eps * (mlp->Layers[i - 1].Neurons[j].OutArcs[k].target->BPerr) * mlp->Layers[i - 1].Neurons[j].output) + beta * mlp->Layers[i - 1].Neurons[j].OutArcs[k].oldDelta;
							mlp->Layers[i - 1].Neurons[j].OutArcs[k].weight += mlp->Layers[i - 1].Neurons[j].OutArcs[k].oldDelta;
							mlp->Layers[i - 1].Neurons[j].absDeltaSum += abs(mlp->Layers[i - 1].Neurons[j].OutArcs[k].oldDelta);
						}
					}
				}

				mlp->resetVectorsProfiler(false, true);

				//calcolo dei vettori di influenza dell'errore retropropagato da ogni output
				for (int i = mlp->nLayers - 2; i >= 0; i--) {
					for (int j = 0; j < mlp->numNeurons(i); j++) {
						for (int k = 0; k < mlp->numCon(i, j); k++) {
							mlp->getTarget(i, j)->influenceOutput[k] += abs((mlp->getTarget(i, j)->OutArcs[k].oldDelta) / (mlp->getTarget(i, j)->absDeltaSum));
						}
					}
				}

				///////////////////////////////////sezione dell'algoritmo in cui la rete ha i vettori di propagazione caricati/////////////////////////

				hpd->trainSupportNet(Wi, We, g);

				///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

				Gerr += Err;
				mlp->resetPotential(); // azzero i potenziali di output della rete
				mlp->resetAbsSumPotenzial(); //azzero le sommatorie in valore assoluto dei contributi delle connessioni nell'output del neurone

				t1 = mlp->PerformanceCounter();
				elapsedMilliseconds = ((t1 - t0) * 1000.0) / mlp->PerformanceFrequency();
				executionTime += elapsedMilliseconds;
			}

			///////////////////////////////////// procedura di eliminazione archi ////////////////////////////////////////////////////

			if (t > cutStart) {
				if (t == cutStart + 1) nWt0 = mlp->nArc;
				WdelP = 1.0f - (mlp->nArc / (float)nWt0);
				WdelP = WdelP * 100.0f;
				if (mlp->NetErrPercent > SmaxErr || WdelP >= maxWloss) {
					if (mlp->NetErrPercent > SmaxErr) cout << "error reached: " << mlp->NetErrPercent << endl;
					if (WdelP >= maxWloss) cout << "weights lossed: " << WdelP << "%" << endl;
					return;
				}
				hpd->cutMadArc(1, alfa);
			}

			//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

			Gerr /= 2;
			Perr /= mlp->examples.size()*Out.size();
			executionTime /= mlp->examples.size();
			inputTime /= mlp->examples.size();
			mlp->NetPerformance = inputTime;
			mlp->NetErrPercent = Perr;



			cout << "Iterazione  " << t << "   errore quadratico: " << Gerr << "   errore percentuale medio:  " << Perr << "  %err" << endl
				<< "exemple time: " << executionTime << " ms  inputTime: " << inputTime << " ms" << endl
				<< "Net integrity:" << (mlp->nArc / (float)nWt0) * 100 << "%    arcs:  " << mlp->nArc << endl;

			//if (t > 100 && t < 200)Sleep(500);
			if (Perr < ErrPercent) {
				cout << "Percentuale di errore obbiettivo raggiunta!" << endl;
				return;
			}

		}
	}

};

////////////////////////////////////////////////////////////////CUDA Kernels//////////////////////////////////////////////////////////

//resetta il valore di una variabile all'interno della scheda grafica
__global__ void CUDAresetVar(float *val) {
	*val = 0;
}
//applica ad ogni arco della rete la correzione del peso
__global__ void CUDAapplyWeightCorrections(float eps, float *NeuronOut, float *BPerr, float *weights, int *ArcIn, int *ArcOut, int nArcs) {
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < nArcs) {
		weights[i] += -eps * BPerr[ArcIn[i]] * NeuronOut[ArcOut[i]];
	}
}
__global__ void CUDAapplyBayesCorrections(float eps, float *BPerr, float *Bayes, int startN, int endN) {
	unsigned int i = startN + (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i <= endN) {
		Bayes[i] += -eps * BPerr[i];
	}
}
//applica alla sommatoria degli errori pesati e retropropagati ad ogni neurone la derivata puntuale della sigmoide 
//DEPRECATED!!!
/*__global__ void CUDAapplayDsigToBPerr(float *NeuronOut, float *BPerr, int nNeuron) {
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < nNeuron) {
		BPerr[i] *= NeuronOut[i] * (1 - NeuronOut[i]);
	}
}*/
//retropropaga l'errore nella rete 
__global__ void CUDAPropagationErr(float *BPerr, float *weights, float *NeuronOut, int *ArcIn, int *ArcOut, int startA, int endA) {
	unsigned int i = startA + (blockIdx.x * blockDim.x) + threadIdx.x;

	//retropropago l'errore dai neuroni successivi
	if (i <= endA) {
		//BPerr[ArcOut[i]] += BPerr[ArcIn[i]] * weights[i];
		atomicAdd(&BPerr[ArcOut[i]], BPerr[ArcIn[i]] * weights[i]);
	}
}
__global__ void CUDAoutDiff(float *BPerr, float *NeuronOut, int startN, int endN) {
	int i = startN + (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i <= endN) {
		BPerr[i] *= NeuronOut[i] * (1 - NeuronOut[i]);
	}
}
//calcola l'errore dei neuroni dello strato output
__global__ void CUDAoutputErr(float *NeuronOut, int OutputRef, int numNeurons, int inputN, float *BPerr, float *examples, int exampleRef, float *mapMaxOut, float *mapMinOut, float *MeanErr) {
	unsigned int i = (OutputRef) + (blockIdx.x * blockDim.x) + threadIdx.x; //indice di scorrimento vettori: NeuronOut, BPerr, 
	unsigned int e = (exampleRef + inputN) + (blockIdx.x * blockDim.x) + threadIdx.x; //indice di scorrimento vettori: examples
	unsigned int m = (blockIdx.x * blockDim.x) + threadIdx.x; // indice di scorrimento vettori: mapMaxOut, mapMinOut
	//if (i == 0) *MeanErr = 0;
	if (i < numNeurons) {

		float delta = mapMaxOut[m] - mapMinOut[m];
		BPerr[i] = (NeuronOut[i] - ((examples[e] - mapMinOut[m]) / delta)) * NeuronOut[i] * (1 - NeuronOut[i]); // formula valida solo per i neuroni di uscita
		//atomicAdd(MeanErr, (abs((((NeuronOut[i] * delta) + mapMinOut[m]) - examples[e]) / examples[e]))*100.0f);
		atomicAdd(MeanErr, abs((((NeuronOut[i] * delta) + mapMinOut[m]) - examples[e])/ examples[e]) * 100.0f);
		//*MeanErr += abs((examples[e] - ((NeuronOut[i] * delta) + mapMinOut[m])) / examples[e]); // calcolo l'errore percentuale sulla singola uscita e lo sommo 
	}
}
//resetta un dato vettore 
__global__ void CUDAresetVector(float *vect, int size) {
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < size) vect[i] = 0.0f;
}
//imposta i valori di output dei neuroni di input al valore dell'esempio
__global__ void CUDAsetInput(float *NeuronOut, int inputN, int exampleRef, float *example) {
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < inputN)NeuronOut[i] = example[exampleRef + i];
}
__global__ void CUDAsetSingleInput(float *NeuronOut, int inputN, float *example) {
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < inputN)NeuronOut[i] = example[i];
}
//applica la sigmoide ai potenziali dei neuroni in un dato intervallo
__global__ void CUDAsigLayer(float *NeuronOut, int start, int end) {
	unsigned int i = start + (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i <= end) {
		NeuronOut[i] = 1 / (1 + expf(-NeuronOut[i]));
	}
}
//aggiunge all'output del neurone il contributo del bayes
__global__ void CUDAbayesInput(float *NeuronOut, float *Bayes, int start, int end) {
	unsigned int i = start + (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i <= end) {
		NeuronOut[i] += Bayes[i];
	}
}
//propaga l'informazione dai neuroni dello strato input a quello di output
//TODO sostituire l'utilizzo di atomicAdd con la riduzione delle somme (utilizando atomicAdd il minor numero possibile di volte per ogni neurone)
__global__ void CUDAlayerInput(float *weights, int *ArcIn, int *ArcOut, float *NeuronOut, int start, int end) {
	unsigned int i = start + (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i <= end) {
		atomicAdd(&NeuronOut[ArcIn[i]], NeuronOut[ArcOut[i]] * weights[i]); //addizione bloccante non permette ad altri thread di sovrascrivere il falore finche l'operazione non � completata
		//printf("Neurone %d ( %f ) += Neuron %d ( %f ) * peso ( %f ) \n", ArcIn[i], NeuronOut[ArcIn[i]], ArcOut[i], NeuronOut[ArcOut[i]], weights[i]);
		//NeuronOut[ArcIn[i]] += NeuronOut[ArcOut[i]] * weights[i];
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

class CUDAcore {
	//api di interfacciamento alla GPU
private:

public:
	//TODO aggiungere il vettore dei bayes e relativa funzione di applicazione e correzione

	hipDeviceProp_t prop; //Device specs struct
	int GpuID = 0;

	//struttura contenente i puntatori alle aree di memoria conenenti i parametri della rete nella GPU
	struct devNetParams {
		float *weights = 0;
		int *ArcIn = 0;
		int *ArcOut = 0;
		float *examples = 0;
		float *NeuronOut = 0;
		float *Bayes = 0;
		float *BPerr = 0;
		float *mapMaxOut = 0;
		float *mapMinOut = 0;
		int *NeurInLyr = 0;
		int *priority = 0;
		float *MeanErr = 0;
		float *InputRT = 0;
	}gpuNetParams;

	vector<float> weights; //pei della rete
	vector<int> ArcIn; //target dell'n-esimo arco
	vector<int> ArcOut; //base dell'n-esimo arco
	vector<float> NeuronOut; //vettore contenente l'output dei neuroni
	vector<float> Bayes; //vettore contenente i bayes dei neuroni
	vector<float> BPerr; // vettore contenete gli errori retropropagati
	vector<float> mapMaxOut; //vettore contenente il massimo valore degli output
	vector<float> mapMinOut; //vettore contenente il minimo valore degli output
	vector<int> priority; // vettore contenente i punti di sincronizazione dei thread
	vector<int> NeurInLyr; //vettore contenente gli indici dell'ultimo neurone di ogni layer
	vector<float>examples; //vettore degli esempi
	float MeanErr = 0; //veriabile contenente l'errore medio percentuale della rete
	int inputN, outputN; //passo di esecuzione elementi del vettore esempi

	CUDAcore(int nGpu) {
		GpuID = nGpu;
		checkCuda(hipGetDeviceProperties(&prop, nGpu)); // carica lo struct hipDeviceProp_t prop con le caratteristiche della GPU con indice 0
	}
	/*per convertire gli oggetti vector in Array
	std::vector<double> v;
	double* a = &v[0];
	*/

	void cudaNetCopyMLP(MLP *pt) {
		cout << "copying the net into CUDAcore.." << endl;
		weights.resize(pt->nArc);
		ArcIn.resize(pt->nArc);
		ArcOut.resize(pt->nArc);
		NeuronOut.resize(pt->nNeurons);
		Bayes.resize(pt->nNeurons);
		BPerr.resize(pt->nNeurons);
		priority.resize(pt->nLayers + 1);
		NeurInLyr.resize(pt->nLayers + 1);
		mapMaxOut.resize(pt->map.size());
		mapMinOut.resize(pt->map.size());
		inputN = pt->numNeurons(0);
		outputN = pt->numNeurons(pt->nLayers - 1);

		int NeuronIdx = 0;
		int ArcIdx = 0;
		vector<int> neurons(pt->nLayers);
		//carico il vettore di mappatura dell'output della rete
		for (int i = 0; i < pt->map.size(); i++) {
			mapMaxOut[i] = pt->map[i].maxValue;
			mapMinOut[i] = pt->map[i].minValue;
		}

		NeurInLyr[0] = -1; // setto il primo valore 
		priority[0] = -1; // setto il primo valore 

		//carico i parametri della rete
		for (int i = 0; i < pt->nLayers; i++) {

			for (int j = 0; j < pt->numNeurons(i); j++) {

				Bayes[NeuronIdx] = pt->getTarget(i, j)->bayes;

				for (int k = 0; k < pt->numCon(i, j); k++) {

					weights[ArcIdx] = pt->getTarget(i, j)->OutArcs[k].weight;
					ArcIn[ArcIdx] = pt->getTarget(i, j)->OutArcs[k].target->neurIdx;
					ArcOut[ArcIdx] = pt->getTarget(i, j)->neurIdx;
					ArcIdx++;
				}
				NeuronIdx++;
			}
			NeurInLyr[i + 1] = NeuronIdx - 1; // salvo l'indice dell'ultimo neurone del layer corrente
			priority[i + 1] = ArcIdx - 1; // salvo l'indice dell'ultimo arco del layer corrente
		}
	}

	void cudaNetPasteMLP(MLP *pt) {
		int idx = 0;
		int Nidx = 0;
		for (int i = 0; i < pt->nLayers; i++) {
			for (int j = 0; j < pt->numNeurons(i); j++) {
				pt->getTarget(i, j)->bayes = Bayes[Nidx++];
				for (int k = 0; k < pt->numCon(i, j); k++) {
					pt->getTarget(i, j)->OutArcs[k].weight = weights[idx++];
				}
			}
		}
	}

	void cudaNetCopyHopfield(Hopfield* pt) {

	}

	void cudaNetCopyExamples(MLP *pt) {
		cout << "copying example into CUDAcore" << endl;
		examples.resize(pt->examples.size()*(pt->numNeurons(0) + pt->numNeurons(pt->nLayers - 1)));
		int idx = 0;
		for (int i = 0; i < pt->examples.size(); i++) {
			for (int j = 0; j < pt->numNeurons(0); j++) {
				examples[idx++] = pt->examples[i].input[j];

			}
			for (int j = 0; j < pt->numNeurons(pt->nLayers - 1); j++) {
				examples[idx++] = pt->examples[i].Doutput[j];
			}
		}
	}
	////////////////////////////////////////////////////////////////CUDA Kernel functions/////////////////////////////////////////////////

	//esegue le operazioni di allocamento memoria e preparazione al lancio del kernel di propagazione della rete
	hipError_t hostCUDAtrainingNet(float eps, int Niter, int ThxBlock) {
		cout << "learning is started!" << endl;
		//host variables
		float *Cweights = &weights[0];
		int *CArcIn = &ArcIn[0];
		int *CArcOut = &ArcOut[0];
		float *CNeuronOut = &NeuronOut[0];
		float *CBayes = &Bayes[0];
		float *CBPerr = &BPerr[0];
		float *CmapMaxOut = &mapMaxOut[0];
		float *CmapMinOut = &mapMinOut[0];
		float *Cexamples = &examples[0];
		int *CNeurInLyr = &NeurInLyr[0];
		int *Cpriority = &priority[0];
		float *CMeanErr = &MeanErr;

		//device variables
		float *dev_weights = 0;
		int *dev_ArcIn = 0;
		int *dev_ArcOut = 0;
		float *dev_examples = 0;
		float *dev_NeuronOut = 0;
		float *dev_Bayes = 0;
		float *dev_BPerr = 0;
		float *dev_mapMaxOut = 0;
		float *dev_mapMinOut = 0;
		int *dev_NeurInLyr = 0;
		int *dev_priority = 0;
		float *dev_MeanErr = 0;

		//int ThxBlock = 1024;

		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(GpuID);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		// Allocate GPU buffers for vectors    
		cudaStatus = hipMalloc((void**)&dev_weights, weights.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_ArcIn, ArcIn.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_ArcOut, ArcOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_NeuronOut, NeuronOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_Bayes, Bayes.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_BPerr, BPerr.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_mapMaxOut, mapMaxOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_mapMinOut, mapMinOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_examples, examples.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_NeurInLyr, NeurInLyr.size() * sizeof(int));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_priority, priority.size() * sizeof(int));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&dev_MeanErr, sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;


		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_weights, Cweights, weights.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_ArcIn, CArcIn, ArcIn.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_ArcOut, CArcOut, ArcOut.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_NeuronOut, CNeuronOut, NeuronOut.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_Bayes, CBayes, Bayes.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_BPerr, CBPerr, BPerr.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_mapMaxOut, CmapMaxOut, mapMaxOut.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_mapMinOut, CmapMinOut, mapMinOut.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_examples, Cexamples, examples.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_NeurInLyr, CNeurInLyr, NeurInLyr.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_priority, Cpriority, priority.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(dev_MeanErr, CMeanErr, sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;



		//////////////////lancio dei kernel all'interno della gpu////////////////
		int startA = 0;
		int endA = 0;
		int startN = 0;
		int endN = 0;
		int numLayerArcs = 0;
		int numLayerNeur = 0;
		int numOfBlocksMax = 0;
		int numOfBlocksA = 0;
		int numOfBlocksN = 0;
		int numOfBlocksOut = floorf(outputN / ThxBlock) + 1;
		int exampleRef = 0;
		int outputRef = NeuronOut.size() - outputN;
		long long t0 = 0, t1 = 0;
		long long t0in = 0, t1in = 0;
		double elapsedMilliseconds = 0;
		double elapsedInMilliseconds = 0;

		//debug////////////
		int en = 0;
		float delta = 0;
		//////////////////

		for (int it = 0; it < Niter; it++) { //scorro le iterazioni

			t0 = PerformanceCounter();

			for (int t = 0; t < (examples.size() / (inputN + outputN)); t++) { //scorro gli esempi
				//imposto il riferimento per l'esempio di input
				exampleRef = t * (inputN + outputN);
				t0in = PerformanceCounter();
				//resetto il vettore contenente lo stato di attivazione dei neuroni
				numOfBlocksA = (floorf(NeuronOut.size() / ThxBlock) + 1);
				CUDAresetVector << <numOfBlocksA, ThxBlock >> > (dev_NeuronOut, NeuronOut.size());

				//////////////////////////////DEBUG///////////////////////////////
				//cudaStatus = hipMemcpy(CNeuronOut, dev_NeuronOut, NeuronOut.size() * sizeof(float), hipMemcpyDeviceToHost);
				//if (cudaCheckStatus(cudaStatus) == true) goto Error;
				//copy(CNeuronOut, CNeuronOut + NeuronOut.size(), NeuronOut.begin());
				///////////////////////////////////////////////////////////////////

				//imposto i valori di input ai neuroni dello strato input
				numOfBlocksA = (floorf(inputN / ThxBlock) + 1);
				CUDAsetInput << <numOfBlocksA, ThxBlock >> > (dev_NeuronOut, inputN, exampleRef, dev_examples);

				//////////////////////////////DEBUG///////////////////////////////
				//cudaStatus = hipMemcpy(CNeuronOut, dev_NeuronOut, NeuronOut.size() * sizeof(float), hipMemcpyDeviceToHost);
				//if (cudaCheckStatus(cudaStatus) == true) goto Error;
				//copy(CNeuronOut, CNeuronOut + NeuronOut.size(), NeuronOut.begin());
				///////////////////////////////////////////////////////////////////

				//propagazione dell'input nella rete

				startA = 0; // indice di partenza dei vettori archi
				endA = 0; // ultimo indice dei vettori archi
				startN = 0; // indice di partenza dei vettori neuroni
				endN = 0; // ultimo indice dei vettori neuroni

				for (int i = 0; i < priority.size() - 1; i++) { //NB non viene applicata la sigmoide allo strato di input eventulmente correggi

					startA = priority[i] + 1;
					endA = priority[i + 1];

					if (i < priority.size() - 2) {
						startN = NeurInLyr[i + 1] + 1;
						endN = NeurInLyr[i + 2];
					}

					numLayerArcs = endA - startA + 1;
					numLayerNeur = endN - startN + 1;

					numOfBlocksA = floorf(numLayerArcs / ThxBlock) + 1;
					numOfBlocksN = floorf(numLayerNeur / ThxBlock) + 1;

					if (i < priority.size() - 2) {
						CUDAlayerInput << <numOfBlocksA, ThxBlock >> > (dev_weights, dev_ArcIn, dev_ArcOut, dev_NeuronOut, startA, endA); //propago l'output dei neuroni al prossimo/i layer
						CUDAbayesInput << < numOfBlocksN, ThxBlock >> > (dev_NeuronOut, dev_Bayes, startN, endN); //applico il contributo dei bayes all output dei neuroni del layer corrente 
						CUDAsigLayer << <numOfBlocksN, ThxBlock >> > (dev_NeuronOut, startN, endN); //applico la sigmoide allo stato di attivazione dei neuroni
						//////////////////////////////DEBUG/////////////////////////////////
						cudaStatus = hipMemcpy(CNeuronOut, dev_NeuronOut, NeuronOut.size() * sizeof(float), hipMemcpyDeviceToHost);
						if (cudaCheckStatus(cudaStatus) == true) goto Error;
						copy(CNeuronOut + outputRef, CNeuronOut + NeuronOut.size(), NeuronOut.begin() + outputRef);
						///////////////////////////////////////////////////////////////////
					}
				}

				t1in = PerformanceCounter();
				elapsedInMilliseconds += ((t1in - t0in) * 1000.0) / PerformanceFrequency();


				//resetto il vettore contenente l'errore  dei neuroni
				numOfBlocksN = (floorf(BPerr.size() / ThxBlock) + 1);
				CUDAresetVector << <numOfBlocksN, ThxBlock >> > (dev_BPerr, BPerr.size());

				//////////////////////////////DEBUG///////////////////////////////
				//cudaStatus = hipMemcpy(CBPerr, dev_BPerr, BPerr.size() * sizeof(float), hipMemcpyDeviceToHost);
				//if (cudaCheckStatus(cudaStatus) == true) goto Error;
				//copy(CBPerr, CBPerr + BPerr.size(), BPerr.begin());
				///////////////////////////////////////////////////////////////////

				CUDAresetVar <<<1, 1 >>> (dev_MeanErr);
				CUDAoutputErr << <numOfBlocksOut, ThxBlock >> > (dev_NeuronOut, outputRef, NeuronOut.size(), inputN, dev_BPerr, dev_examples, exampleRef, dev_mapMaxOut, dev_mapMinOut, dev_MeanErr);
				hipMemcpy(CMeanErr, dev_MeanErr, sizeof(float), hipMemcpyDeviceToHost);
				//////////////////////////////DEBUG///////////////////////////////
				//cudaStatus = hipMemcpy(CBPerr, dev_BPerr, BPerr.size() * sizeof(float), hipMemcpyDeviceToHost);
				//if (cudaCheckStatus(cudaStatus) == true) goto Error;
				//copy(CBPerr, CBPerr + BPerr.size(), BPerr.begin());

				///////////////////////////////////////////////////////////////////

				////////////////////////////////////visualizzazione dell'esempio///////////////////////////////////
				if (en == t) {

					cudaStatus = hipMemcpy(CNeuronOut, dev_NeuronOut, NeuronOut.size() * sizeof(float), hipMemcpyDeviceToHost);
					if (cudaCheckStatus(cudaStatus) == true) goto Error;
					copy(CNeuronOut, CNeuronOut + NeuronOut.size(), NeuronOut.begin());

					cout << "esempio " << en << endl;
					for (int on = 0; on < outputN; on++) {
						delta = mapMaxOut[on] - mapMinOut[on];
						cout << "Y" << on << ": " << (NeuronOut[NeuronOut.size() - outputN + on] * delta) + mapMinOut[on] << "   D" << on << ": " << examples[exampleRef + inputN + on] << endl;
					}
					cout << endl;
					en--;
					if (en < 0)en = (examples.size() / (inputN + outputN)) - 1;
				}
				///////////////////////////////////////////////////////////////////////////////////////////////////

				MeanErr += *CMeanErr / outputN;


				//retropropagazione dell'errore

				for (int i = priority.size() - 2; i > 1; i--) {

					startA = priority[i - 1] + 1;
					endA = priority[i];
					startN = NeurInLyr[i - 1] + 1;
					endN = NeurInLyr[i];

					numLayerArcs = endA - startA + 1;
					numLayerNeur = endN - startN + 1;

					numOfBlocksA = floorf(numLayerArcs / ThxBlock) + 1;
					numOfBlocksN = floorf(numLayerNeur / ThxBlock) + 1;
					//numOfBlocksMax = maxOf(numOfBlocksA, numOfBlocksN);

					CUDAPropagationErr <<<numOfBlocksA, ThxBlock >>> (dev_BPerr, dev_weights, dev_NeuronOut, dev_ArcIn, dev_ArcOut, startA, endA);
					CUDAoutDiff <<<numOfBlocksN, ThxBlock >>> (dev_BPerr, dev_NeuronOut, startN, endN);
					cudaStatus = hipMemcpy(CBPerr, dev_BPerr, BPerr.size() * sizeof(float), hipMemcpyDeviceToHost);
					if (cudaCheckStatus(cudaStatus) == true) goto Error;
					copy(CBPerr, CBPerr + BPerr.size(), BPerr.begin());
				}

				//applico a ogni peso la sua correzione

				startN = NeurInLyr[1] + 1; // la correzione dei bais va applicata dal primo layer nascosto in poi
				endN = NeurInLyr[NeurInLyr.size() - 1];

				numLayerNeur = endN - startN + 1;

				numOfBlocksA = floorf(weights.size() / ThxBlock) + 1;
				numOfBlocksN = floorf(numLayerNeur / ThxBlock) + 1;

				CUDAapplyWeightCorrections << <numOfBlocksA, ThxBlock >> > (eps, dev_NeuronOut, dev_BPerr, dev_weights, dev_ArcIn, dev_ArcOut, weights.size());
				CUDAapplyBayesCorrections << <numOfBlocksN, ThxBlock >> > (eps, dev_BPerr, dev_Bayes, startN, endN);

				////////////////////DEBUG SECTION////////////////////////
				//cudaStatus = hipMemcpy(Cweights, dev_weights, weights.size() * sizeof(float), hipMemcpyDeviceToHost);
				//if (cudaCheckStatus(cudaStatus) == true) goto Error;
				//copy(Cweights, Cweights + weights.size(), weights.begin());

				//cudaStatus = hipMemcpy(CBayes, dev_Bayes, Bayes.size() * sizeof(float), hipMemcpyDeviceToHost);
				//if (cudaCheckStatus(cudaStatus) == true) goto Error;
				//copy(CBayes, CBayes + Bayes.size(), Bayes.begin());

				//cudaStatus = hipMemcpy(CBPerr, dev_BPerr, BPerr.size() * sizeof(float), hipMemcpyDeviceToHost);
				//if (cudaCheckStatus(cudaStatus) == true) goto Error;
				//copy(CBPerr, CBPerr + BPerr.size(), BPerr.begin());
				/////////////////////////////////////////////////////////

			}

			t1 = PerformanceCounter();
			elapsedMilliseconds = ((t1 - t0) * 1000.0) / PerformanceFrequency(); // calcolo il tempo di esecuzione di una iterazione di addestramento (tutto il set)
			MeanErr = MeanErr / (examples.size() / (inputN + outputN)); //calcolo l'errore percentuale medio sul dataset
			elapsedInMilliseconds = elapsedInMilliseconds / (examples.size() / (inputN + outputN));
			cout << "Iterazione: " << it << "  " << MeanErr << " %Err  " << "execution time:" << elapsedMilliseconds << "ms" << endl;
			cout << "mean InputTime: " << elapsedInMilliseconds << "ms" << endl;
			printNetSpecs();
			MeanErr = 0;
		}

		cudaStatus = hipMemcpy(Cweights, dev_weights, weights.size() * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		copy(Cweights, Cweights + weights.size(), weights.begin());

		cudaStatus = hipMemcpy(CBayes, dev_Bayes, Bayes.size() * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		copy(CBayes, CBayes + Bayes.size(), Bayes.begin());
		//checkpoint di errore (se la GPU richiama un qualunque errore ripare da qui)
	Error:

		//libero la memoria nella scheda grafica
		hipFree(dev_weights);
		hipFree(dev_ArcIn);
		hipFree(dev_ArcOut);
		hipFree(dev_NeuronOut);
		hipFree(dev_examples);
		hipFree(dev_BPerr);
		hipFree(dev_mapMaxOut);
		hipFree(dev_mapMinOut);
		hipFree(dev_priority);
		hipFree(dev_NeurInLyr);

		//ritorno lo stato della GPU
		return cudaStatus;
	}

	//esegue il caricamento nella gpu dei parametri della rete
	hipError_t hostCUDAuploadNetParams() {

		hipError_t cudaStatus;

		cudaStatus = hipSetDevice(GpuID);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		//host variables
		float *Cweights = &weights[0];
		int *CArcIn = &ArcIn[0];
		int *CArcOut = &ArcOut[0];
		float *CNeuronOut = &NeuronOut[0];
		float *CBayes = &Bayes[0];
		float *CBPerr = &BPerr[0];
		float *CmapMaxOut = &mapMaxOut[0];
		float *CmapMinOut = &mapMinOut[0];
		float *Cexamples = &examples[0];
		int *CNeurInLyr = &NeurInLyr[0];
		int *Cpriority = &priority[0];
		float *CMeanErr = &MeanErr;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(GpuID);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		// Allocate GPU buffers for vectors    
		cudaStatus = hipMalloc((void**)&gpuNetParams.weights, weights.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.ArcIn, ArcIn.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.ArcOut, ArcOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.NeuronOut, NeuronOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.Bayes, Bayes.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.BPerr, BPerr.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.mapMaxOut, mapMaxOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.mapMinOut, mapMinOut.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.examples, examples.size() * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.NeurInLyr, NeurInLyr.size() * sizeof(int));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.priority, priority.size() * sizeof(int));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.InputRT, inputN * sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMalloc((void**)&gpuNetParams.MeanErr, sizeof(float));
		if (cudaCheckStatus(cudaStatus) == true) goto Error;


		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(gpuNetParams.weights, Cweights, weights.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.ArcIn, CArcIn, ArcIn.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.ArcOut, CArcOut, ArcOut.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.NeuronOut, CNeuronOut, NeuronOut.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.Bayes, CBayes, Bayes.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.BPerr, CBPerr, BPerr.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.mapMaxOut, CmapMaxOut, mapMaxOut.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.mapMinOut, CmapMinOut, mapMinOut.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.examples, Cexamples, examples.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.NeurInLyr, CNeurInLyr, NeurInLyr.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.priority, Cpriority, priority.size() * sizeof(int), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
		cudaStatus = hipMemcpy(gpuNetParams.MeanErr, CMeanErr, sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		if (false) {
			Error:
			//libero la memoria nella scheda grafica
			hipFree(gpuNetParams.weights);
			hipFree(gpuNetParams.ArcIn);
			hipFree(gpuNetParams.ArcOut);
			hipFree(gpuNetParams.NeuronOut);
			hipFree(gpuNetParams.examples);
			hipFree(gpuNetParams.BPerr);
			hipFree(gpuNetParams.mapMaxOut);
			hipFree(gpuNetParams.mapMinOut);
			hipFree(gpuNetParams.priority);
			hipFree(gpuNetParams.NeurInLyr);
			cout << "ERRORE: libero la memoria della gpu. " << endl;
		}
			
		return cudaStatus;
	}

	//esegue il download dalla gpu dei parametri della rete
	hipError_t hostCUDAdownloadNetParams() {

		cout << "downloading net params from gpu.." << endl;

		float *Cweights = &weights[0];
		float *CBayes = &Bayes[0];

		hipError_t cudaStatus;

		cudaStatus = hipSetDevice(GpuID);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		cudaStatus = hipMemcpy(Cweights, gpuNetParams.weights, weights.size() * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		cudaStatus = hipMemcpy(CBayes, gpuNetParams.Bayes, Bayes.size() * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;


		if (false) {
		Error:
			//libero la memoria nella scheda grafica
			hipFree(gpuNetParams.weights);
			hipFree(gpuNetParams.ArcIn);
			hipFree(gpuNetParams.ArcOut);
			hipFree(gpuNetParams.NeuronOut);
			hipFree(gpuNetParams.examples);
			hipFree(gpuNetParams.BPerr);
			hipFree(gpuNetParams.mapMaxOut);
			hipFree(gpuNetParams.mapMinOut);
			hipFree(gpuNetParams.priority);
			hipFree(gpuNetParams.NeurInLyr);
			cout << "ERRORE: libero la memoria della gpu. " << endl;
		}

		return cudaStatus;
	}

	//esegue l'input della rete gia addestrata prendendo in input l'esempio dato 
	hipError_t hostCUDAInputNet(float *input, int ThxBlock) {
		//inportante verificare che l'input abbia la stessa dimansione dell'input della rete

		hipError_t cudaStatus;

		cudaStatus = hipSetDevice(GpuID);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		//////////////////lancio dei kernel all'interno della gpu////////////////
		//int ThxBlock = 1024;
		int startA = 0;
		int endA = 0;
		int startN = 0;
		int endN = 0;
		int numLayerArcs = 0;
		int numLayerNeur = 0;
		int numOfBlocksMax = 0;
		int numOfBlocksA = 0;
		int numOfBlocksN = 0;
		int numOfBlocksOut = floorf(outputN / ThxBlock) + 1;
		int outputRef = NeuronOut.size() - outputN;
		long long t0in = 0, t1in = 0;
		double elapsedInMilliseconds = 0;
			
		t0in = PerformanceCounter();
		//resetto il vettore contenente lo stato di attivazione dei neuroni
		numOfBlocksA = (floorf(NeuronOut.size() / ThxBlock) + 1);
		CUDAresetVector <<<numOfBlocksA, ThxBlock >>> (gpuNetParams.NeuronOut, NeuronOut.size());

		cudaStatus = hipMemcpy(gpuNetParams.InputRT, input, inputN * sizeof(float), hipMemcpyHostToDevice);
		if (cudaCheckStatus(cudaStatus) == true) goto Error;

		//imposto i valori di input ai neuroni dello strato input
		numOfBlocksA = (floorf(inputN / ThxBlock) + 1);
		CUDAsetSingleInput <<<numOfBlocksA, ThxBlock>>> (gpuNetParams.NeuronOut, inputN, gpuNetParams.InputRT);
		
		//propagazione dell'input nella rete

		startA = 0; // indice di partenza dei vettori archi
		endA = 0; // ultimo indice dei vettori archi
		startN = 0; // indice di partenza dei vettori neuroni
		endN = 0; // ultimo indice dei vettori neuroni

		for (int i = 0; i < priority.size() - 1; i++) { //NB non viene applicata la sigmoide allo strato di input eventulmente correggi

			startA = priority[i] + 1;
			endA = priority[i + 1];

			if (i < priority.size() - 2) {
				startN = NeurInLyr[i + 1] + 1;
				endN = NeurInLyr[i + 2];
			}

			numLayerArcs = endA - startA + 1;
			numLayerNeur = endN - startN + 1;

			numOfBlocksA = floorf(numLayerArcs / ThxBlock) + 1;
			numOfBlocksN = floorf(numLayerNeur / ThxBlock) + 1;

			if (i < priority.size() - 2) {
				CUDAlayerInput <<<numOfBlocksA, ThxBlock >>> (gpuNetParams.weights, gpuNetParams.ArcIn, gpuNetParams.ArcOut, gpuNetParams.NeuronOut, startA, endA); //propago l'output dei neuroni al prossimo/i layer
				CUDAbayesInput <<<numOfBlocksN, ThxBlock >>> (gpuNetParams.NeuronOut, gpuNetParams.Bayes, startN, endN); //applico il contributo dei bayes all output dei neuroni del layer corrente 
				CUDAsigLayer <<<numOfBlocksN, ThxBlock >>> (gpuNetParams.NeuronOut, startN, endN); //applico la sigmoide allo stato di attivazione dei neuroni
				
			}
		}

		//copio l'output dei neuroni dello strato output nella memoria della cpu
		cudaStatus = hipMemcpy(&NeuronOut[0] + outputRef, gpuNetParams.NeuronOut + outputRef, outputN * sizeof(float), hipMemcpyDeviceToHost); //TODO da errore e non carica il vettore trovare il BUG
		if (cudaCheckStatus(cudaStatus) == true) goto Error;
			
		t1in = PerformanceCounter();
		elapsedInMilliseconds = ((t1in - t0in) * 1000.0) / PerformanceFrequency();

		////////////////////////////////////visualizzazione dell'esempio///////////////////////////////////

		float delta;
		cout << "input time: " << elapsedInMilliseconds << " ms" << endl;

		for (int on = 0; on < outputN; on++) {
			delta = mapMaxOut[on] - mapMinOut[on];
			cout << "Y" << on << ": " << (NeuronOut[NeuronOut.size() - outputN + on] * delta) + mapMinOut[on] << endl;
		}
		cout << endl;
		
		///////////////////////////////////////////////////////////////////////////////////////////////////

		if (false) {
		Error:
			//libero la memoria nella scheda grafica
			hipFree(gpuNetParams.weights);
			hipFree(gpuNetParams.ArcIn);
			hipFree(gpuNetParams.ArcOut);
			hipFree(gpuNetParams.NeuronOut);
			hipFree(gpuNetParams.examples);
			hipFree(gpuNetParams.BPerr);
			hipFree(gpuNetParams.mapMaxOut);
			hipFree(gpuNetParams.mapMinOut);
			hipFree(gpuNetParams.priority);
			hipFree(gpuNetParams.NeurInLyr);
		}

		return cudaStatus;
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////CUDA UTILITY//////////////////////////////////////////////////////////////////
	//verifica la corretta esecuzione di un operazione
	inline hipError_t checkCuda(hipError_t result)
	{
#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			assert(result == hipSuccess);
		}
#endif
		return result;
	}
	//verifica la corretta esecuzione di un operazione restituendo un bool
	bool cudaCheckStatus(hipError_t cudaStatus) {
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			return true;
		}
	}
	//stampa a schermo le principali propriet� della scheda
	void printDeviceSpecs() {

		printf("\nDevice: %s\n", prop.name);
		printf("Cores clock: %d MHz\n", (prop.clockRate / 1000));
		printf("Memory clock: %d MHz\n", (prop.memoryClockRate / 1000));
		printf("Total global memmory %.2f MB\n", (float)(prop.totalGlobalMem / (1024 * 1024)));
		printf("Max grid size: x %d, y %d, z %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Max block axis size: x %d, y %d, z %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Warp size: %d\n", prop.warpSize);
		printf("Max therads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max therads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
		printf("Compute Mode: %d\n", prop.computeMode);
		printf("Host mem access: %d\n", prop.canUseHostPointerForRegisteredMem);
		printf("Shared mem per multiprocessor %.2f KB\n", (float)(prop.sharedMemPerMultiprocessor / 1024));
		printf("Max shared mem per blocco %.2f KB\n", (float)(prop.sharedMemPerBlock / 1024));

	}
	//stampa i parametri della rete che vengono passati alla scheda
	void printNetSpecs() {
		cout << "dimensione del modello: " << sizeOfModel("MB") << " MB" << endl;
		cout << "numero totale dei neuroni : " << NeuronOut.size() << "(" << sizeOfVector(NeuronOut, "KB") + sizeOfVector(Bayes, "KB") + sizeOfVector(BPerr, "KB")<< " KB)" << endl;
		cout << "numero totale degli archi : " << weights.size() << "(" << sizeOfVector(weights, "MB") + sizeOfVector(ArcIn, "MB") + sizeOfVector(ArcOut, "MB") << " MB)" << endl;
		cout << "numero esempi : " << examples.size() / (inputN + outputN) << "  (" << sizeOfVector(examples, "MB") << " MB)" << endl;
	}
	//calcola il peso del modello
	float sizeOfModel(string mesureUnit = "B") {
		int size = 0;
		int scale = 0;
		size += sizeof(weights[0])*weights.size(); //dimensione del vettore pesi
		size += sizeof(ArcIn[0])*ArcIn.size(); //dimensione del vettore contenente i target degli archi
		size += sizeof(ArcOut[0])*ArcOut.size(); //dimensione del vettore contenete i neuroni base degli archi
		size += sizeof(NeuronOut[0])*NeuronOut.size(); //dimensione del vettore contenente gli output dei neuroni
		size += sizeof(Bayes[0])*Bayes.size(); //dimensione del vettore contenente i bayes
		size += sizeof(BPerr[0])*BPerr.size(); //dimensione del vettore contenente 
		size += sizeof(priority[0])*priority.size(); // dimensione del vettore priorit�
		size += sizeof(examples[0])*examples.size(); //dimensione del vettore di esmpio
		if (mesureUnit == "B") { scale = 1; } //Byte
		else if (mesureUnit == "KB") { scale = 1024; } //Kilobyte
		else if (mesureUnit == "MB") { scale = 1024 * 1024; } // Megabyte
		else if (mesureUnit == "GB") { scale = 1024 * 1024 * 1024; } //Gigabyte
		else { cout << "L'unit� di misura non � corretta!!" << endl; return 0.0f; }
		return (float)(size / scale);
	}
	template<typename T, typename A>
	float sizeOfVector(vector<T, A> const& vect,string mesureUnit = "B") {
		int size = 0;
		int scale = 0;
		size = sizeof(vect[0])*vect.size(); //dimensione del vettore di esmpio
		if (mesureUnit == "B") { scale = 1; } //Byte
		else if (mesureUnit == "KB") { scale = 1024; } //Kilobyte
		else if (mesureUnit == "MB") { scale = 1024 * 1024; } // Megabyte
		else if (mesureUnit == "GB") { scale = 1024 * 1024 * 1024; } //Gigabyte
		else { cout << "L'unit� di misura non � corretta!!" << endl; return 0.0f; }
		return (float)((float)size /(float)scale);
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////ALTRE FUNZIONI////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////WINDOWS HIGH SPEED TIMING//////////////////////////////////////////////////////
	BOOL WINAPI QueryPerformanceCounter(_Out_ LARGE_INTEGER *lpPerformanceCount);
	BOOL WINAPI QueryPerformanceFrequency(_Out_ LARGE_INTEGER *lpFrequency);
	inline long long PerformanceCounter() noexcept
	{
		LARGE_INTEGER li;
		::QueryPerformanceCounter(&li);
		return li.QuadPart;
	}
	inline long long PerformanceFrequency() noexcept
	{
		LARGE_INTEGER li;
		::QueryPerformanceFrequency(&li);
		return li.QuadPart;
	}
	/* HOW TO USE:
	long long t0 = PerformanceCounter();
	//code to bench..
	long long t1 = PerformanceCounter();
	double elapsedMilliseconds = ((t1 - t0) * 1000.0) / PerformanceFrequency();
	*/
	//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
};


class genEvolve {
private:
	vector<MLP> mlps; // vettore di reti mlp
	vector<Hopfield> hpds; // vettore di reti Hopfield
public:

};


int main()
{
	CUDAcore gtx760(0);
	gtx760.printDeviceSpecs();

	int layer = 10;
	int columns = 50;
	int out = 4;
	int in = 9;

	MLP a("cudaTest");
	//a.getNetParams();
	a.qubeNet(layer, columns, in, out, true, 0.0001f);
	a.setNetMap(800, 0);
	//a.genTestDataset(500, in, out, 0.1, 3, 0);
	a.getDataset("datnc");
	a.datasetOffset(200.0f);
	//float test[] = {2, 2, 2, 2, 2, 2};
	//a.BP(5, 0.0000000001f, 0, 1);
	gtx760.cudaNetCopyMLP(&a);
	gtx760.cudaNetCopyExamples(&a);
	
	//da usare senza " gtx760.hostCUDAtrainingNet "
	//gtx760.hostCUDAuploadNetParams();
	//gtx760.hostCUDAInputNet(test, 1024);
	//gtx760.hostCUDAdownloadNetParams();

	gtx760.hostCUDAtrainingNet(10.0e-10f, 200, 256);
	gtx760.cudaNetPasteMLP(&a);

	a.saveNet("datncNet-l(5)-c(60)-FC");

	a.BP(5, 10.0e-10f, 0, 1);

	a.saveNet("CpuTest5");
	/*
	system("pause");
	int layer = 3;
	int columns = 50;
	int out = 7;
	int in = 7;
	//MLP a("GenomaX2");
	MLP b("GenomaX2");
	//b.getNetParams();
	b.qubeNetFC(layer, columns, in, out, false);

	//DatasetCore data;
	//data.readTimeSeriesCsv("LottoHistory", out, in/out, 100);
	//b.examples = data.getDataset(0);
	//b.saveDataset("LotoDatatset");

	//b.getDataset("LotoDatatset");
	//vector<int> dims = {in, columns, out};
	//b.customNet(layer, dims, 1);

	Hopfield a("genoma4", &b);
	StructuralLearning SL(&b, &a);

	//a.saveNet("GenomaX");
	////a.toroidNet(layer, dims, 0.3);
	////a.supportNet(1);
	b.setNetMap(4, 0);

	//cout << "caricamento eseguito" << endl;
	b.genTestDataset(50, in, out, 0.2, 3, 2);
	b.BP(200, 0.0001, 0.2, 5);
	//SL.StructuralBP(200, 0.2, 0.3, 0.4, 0.001, -0.001, 8, 5, 0.05, 3, 40);
	////a.getDataset("Dataset");
	////cout << "dataset genrato" << endl;

	//b.BP(200, 0.2, 0.3, 0.9);
	////cout << "addestramento completato" << endl;
	//a.saveNet("GenomaX5");
	//b.saveNet("GenomaX");
	b.saveNet("GenomaX2");
	//cout << "salvataggio eseguito" << endl;
	////system("pause");
	*/
	return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/*NOTE:
	_______________________________________________________________________________________________________________________________
	TASKS COMPLETATE:

	-Hopfield::suportnet() ora crea una struttura su una sola linea con le connessioni nello stesso ordine e verso con i rispettvi
		neuroni delle due reti

	-fuzioni aggiornate per il cambio di puntamento dei vettori di propagazione:
		stampInputInfluences(), stampOutputErrorPropagation(), initVectorsProfiler(), resetVectorsProfiler(), StructuralBP(), inputNetProfiler()
	_______________________________________________________________________________________________________________________________

	TASKS TODO:

	-creare delle funzioni di eliminazione arco e neurone apposite per la struttura mlp + supportNet + binds nella classe Hopfield
		e rimettere aposto le funzioni di eliminazione arco e neurone nella classe network

	-rivedere la funzione NetInputProfiler() nella procedura per il calcolo dell' influnza dell'input (procedura inefficiente)

	-determinare una funzione che possa essere utilizzata anche per Hopfield::AssociativeCorrelation()

	-scrivere la funzione di apprendimento della rete hopfield di supporto

	-aggiungere la propagazione temporle pesata dell'output del neurone

	-verificare che non ci siano bug nella funzione trainSupportNet() (possibile scambio di indici)

	-nella funzione taglia archi si possono verificare dei problemi eliminando degli archi che eliminano neuroni a catena
		il problema � stato risolto per il taglio indiretto di un neurone ma non per piu consecutivi per i quali non viene
		eliminato il bind e probabilmente alcuni parametri non vengono modificati

	-La funzione hostCUDAtrainingNet() � affetta da RACE CONDITIONS rivedere l'algoritmo per minimizare l'utilizzo
		delle Atomic functions .. la correzione va estesa alle funzioni che costruiscono la struttura linearizata della rete da
		passare alla GPU
	________________________________________________________________________________________________________________________________

	POSSIBILI PATCH:

	*/